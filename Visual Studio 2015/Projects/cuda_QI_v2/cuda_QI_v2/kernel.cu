#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <conio.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>  


#define PI 3.141592653
#define BLK_DIM 32
#define CHECK(call)  \
     { const hipError_t error = call;         \
       if (error != hipSuccess) {            \
            printf("Error:%s:%d, ",__FILE__,__LINE__);  \
            printf("code:%d, reason:%s\n",error,hipGetErrorString(error)); \
            exit(1);     \
          }}

int compInc(const void *a, const void *b)
{
	return *(int *)a - *(int *)b;
}

__global__ void compute_radial_profile
(float *img_array, float xc, float yc, float *r_sample, float *t_sample, int t_size, int r_size, float *radial_profile, int arraySize)
{
	const unsigned int idx = (blockIdx.x*blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y*blockDim.y) + threadIdx.y;
	int thread_idx = ((gridDim.x*blockDim.x)*idy) + idx;
	//int i = thread_idx / (arraySize*arraySize);

	//int inner_thread_idx = thread_idx % (arraySize*arraySize);
	int id_r = thread_idx / t_size;
	int id_t = thread_idx % t_size;
	float x = xc + r_sample[id_r] * sin(t_sample[id_t]);
	float y = yc + r_sample[id_r] * cos(t_sample[id_t]);
	int x1 = (int)x;
	int y1 = (int)y;
	int x2 = x1 + 1;
	int y2 = y1 + 1;
	float rdp = img_array[y1*arraySize + x1] * ((float)x2 - x)*((float)y2 - y) + \
		img_array[y2*arraySize + x1] * ((float)x2 - x)*(y - (float)y1) + \
		img_array[y1*arraySize + x2] * (x - (float)x1)*((float)y2 - y) + \
		img_array[y2*arraySize + x2] * (x - (float)x1)*(y - (float)y1);
	float power_id = (float)(id_r) / r_size;

	float fall = 1.0 - exp(-(1.0 - power_id)*(1.0 - power_id) / 0.05);
	float rise = 1.0 - exp(-power_id*power_id / 0.01);
	float power = sqrt(fall*rise*power_id * 2.0);
	radial_profile[thread_idx] = rdp*power;

}

__global__ void compute_radial_profile_MN
(float *img_array, int n_frame, int n_stream, float *xc, float *yc, float *r_sample, float *t_sample, int t_size, int r_size, float *radial_profile, int arraySize)
{
	const unsigned int idx = (blockIdx.x*blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y*blockDim.y) + threadIdx.y;
	int thread_idx = ((gridDim.x*blockDim.x)*idy) + idx;
	//if (thread_idx > (t_size*r_size*n_frame*n_stream))
	//	return;
	int s = (thread_idx+1) / (r_size*t_size);  
	int img_offset = s*arraySize*arraySize;
	int inner_thread_idx = thread_idx % (r_size*t_size);
	int id_r = inner_thread_idx / t_size;
	int id_t = inner_thread_idx % t_size;
	float x = xc[s] + r_sample[id_r] * cos(t_sample[id_t]);
	float y = yc[s] + r_sample[id_r] * sin(t_sample[id_t]);
	int x1 = (int)x;
	int y1 = (int)y;
	int x2 = x1 + 1;
	int y2 = y1 + 1;
	float rdp =
		img_array[y1*arraySize + x1 + img_offset] * ((float)x2 - x)*((float)y2 - y) + \
		img_array[y2*arraySize + x1 + img_offset] * ((float)x2 - x)*(y - (float)y1) + \
		img_array[y1*arraySize + x2 + img_offset] * (x - (float)x1)*((float)y2 - y) + \
		img_array[y2*arraySize + x2 + img_offset] * (x - (float)x1)*(y - (float)y1);
	float power_id = (float)(id_r) / r_size;

	float fall = 1.0 - exp(-(1.0 - power_id)*(1.0 - power_id) / 0.05);
	float rise = 1.0 - exp(-power_id*power_id / 0.01);
	float power = sqrt(fall*rise*power_id * 2.0);
	radial_profile[thread_idx] = rdp*power;
	//d_check[thread_idx] = thread_idx;
}



void getMean(float *img_mat, float *arrayMed, int len, int n_stream)
{
	for (int j = 0; j < n_stream; j++)
	{
		arrayMed[j] = 0;
		for (int i = j*len; i < (j + 1)*len; i++)
		{
			arrayMed[j] += img_mat[i];
		}
		arrayMed[j] = arrayMed[j] / len;
	}

}


void getMean_MN(float *img_mat, float *arrayMed, int len, int n_frame, int n_stream)
{
	for (int k = 0; k < n_stream; k++)
	{
		int s_offset = k*len*n_frame;

		for (int j = 0; j < n_frame; j++)
		{
			int offset = len*j;
			arrayMed[j + k*n_frame] = 0;
			for (int i = 0; i < len; i++)
			{
				arrayMed[j + k*n_frame] += img_mat[i + offset + s_offset];
			}
			arrayMed[j + k*n_frame] = arrayMed[j + k*n_frame] / len;
		}
	}
}


void getrdp_x_y(float *rdp_matrix, int r_N, int theta_num_perQ, float *rdp_all, float *rdp_x, float *rdp_y)
{
	size_t r_byte = sizeof(float)*r_N;
	float *rdp_1 = (float *)malloc(r_byte);
	float *rdp_2 = (float *)malloc(r_byte);
	float *rdp_3 = (float *)malloc(r_byte);
	float *rdp_4 = (float *)malloc(r_byte);

	for (int i = 0; i < r_N; i++)
	{
		rdp_1[i] = 0;
		rdp_2[i] = 0;
		rdp_3[i] = 0;
		rdp_4[i] = 0;

		for (int j = 0; j < theta_num_perQ; j++)
		{
			rdp_1[i] = rdp_1[i] + rdp_matrix[j + 4 * i*theta_num_perQ];

			rdp_2[i] = rdp_2[i] + rdp_matrix[j + 1 * theta_num_perQ + 4 * i*theta_num_perQ];

			rdp_3[i] = rdp_3[i] + rdp_matrix[j + 2 * theta_num_perQ + 4 * i*theta_num_perQ];

			rdp_4[i] = rdp_4[i] + rdp_matrix[j + 3 * theta_num_perQ + 4 * i*theta_num_perQ];
		};
		rdp_all[i] = (rdp_1[i] + rdp_2[i] + rdp_3[i] + rdp_4[i]) / (4 * theta_num_perQ);
	};

	for (int k = 0; k < (2 * r_N - 1); k++)
	{
		if (k < r_N - 1)
		{
			rdp_x[k] = (rdp_2[r_N - 1 - k] + rdp_3[r_N - 1 - k]) / theta_num_perQ;
			rdp_y[k] = (rdp_3[r_N - 1 - k] + rdp_4[r_N - 1 - k]) / theta_num_perQ;
		}
		else
		{
			rdp_x[k] = (rdp_1[k + 1 - r_N] + rdp_4[k + 1 - r_N]) / theta_num_perQ;
			rdp_y[k] = (rdp_1[k + 1 - r_N] + rdp_2[k + 1 - r_N]) / theta_num_perQ;
		}
	}

	free(rdp_1);
	rdp_1 = NULL;
	free(rdp_2);
	rdp_2 = NULL;
	free(rdp_3);
	rdp_3 = NULL;
	free(rdp_4);
	rdp_4 = NULL;


}

void get_rev_rdp(float *rdp, float *rdp_rev, int rdp_x_size)
{
	for (int i = 0; i < rdp_x_size; i++)
	{
		rdp_rev[i] = rdp[rdp_x_size - 1 - i];
	}
	return;
}

void rdp_corr(float *rdp, float *rdp_rev, float *rdp_corr, int N)
{
	float corr_ij;
	int    delay, i, j;

	for (delay = -N + 1; delay < N; delay++)
	{
		//Calculate the numerator
		corr_ij = 0;
		for (i = 0; i < N; i++)
		{
			j = i + delay;
			if ((j < 0) || (j >= N))  //The series are no wrapped,so the value is ignored
				continue;
			else
				corr_ij += (rdp[i] * rdp_rev[j]);
		}

		//Calculate the correlation series at "delay"
		rdp_corr[delay + N - 1] = corr_ij;
	}
}

float findpeak(float *rdp_corr, int N)
{
	float max = rdp_corr[0];
	int index = 0;
	for (int i = 0; i < N; i++)
	{
		if (max <= rdp_corr[i])
		{
			index = i;
			max = rdp_corr[i];
		}
		;
	}
	return index;
}

float LeastSquareGuassian(float *arr_x, float *arr_y, int arr_N)
{
	const int rank_ = 2;
	float atemp[2 * (rank_ + 1)] = { 0 }, b[rank_ + 1] = { 0 }, a[rank_ + 1][rank_ + 1];
	int i, j, k;

	for (i = 0; i < arr_N; i++) {  //
		atemp[1] += arr_x[i];
		atemp[2] += pow(arr_x[i], 2);
		atemp[3] += pow(arr_x[i], 3);
		atemp[4] += pow(arr_x[i], 4);
		//atemp[5] += pow(arr_x[i], 5);
		//atemp[6] += pow(arr_x[i], 6);
		b[0] += arr_y[i];
		b[1] += arr_x[i] * arr_y[i];
		b[2] += pow(arr_x[i], 2) * arr_y[i];
		//b[3] += pow(arr_x[i], 3) * arr_y[i];
	}

	atemp[0] = arr_N;
	/*
	for(i = 0; i <= 2 * rank_; i++)  printf("atemp[%d] = %f\n", i, atemp[i]);
	printf("\n");
	for(i = 0; i <= rank_; i++)  printf("b[%d] = %f\n", i, b[i]);
	printf("\n");
	*/
	for (i = 0; i < rank_ + 1; i++) {
		k = i;
		for (j = 0; j < rank_ + 1; j++)  a[i][j] = atemp[k++];
	}
	/*
	for(i = 0; i < rank_ + 1; i++){
	for(j = 0; j < rank_ + 1; j++)  printf("a[%d][%d] = %-17f  ", i, j, a[i][j]);
	printf("\n");
	}
	printf("\n");
	*/


	for (k = 0; k < rank_ + 1 - 1; k++) {
		int column = k;
		float mainelement = a[k][k];

		for (i = k; i < rank_ + 1; i++)
			if (fabs(a[i][k]) > mainelement) {
				mainelement = fabs(a[i][k]);
				column = i;
			}
		for (j = k; j < rank_ + 1; j++)
		{
			float atemp = a[k][j];
			a[k][j] = a[column][j];
			a[column][j] = atemp;
		}
		float btemp = b[k];
		b[k] = b[column];
		b[column] = btemp;

		for (i = k + 1; i < rank_ + 1; i++) {
			float Mik = a[i][k] / a[k][k];
			for (j = k; j < rank_ + 1; j++)  a[i][j] -= Mik * a[k][j];
			b[i] -= Mik * b[k];
		}
	}
	/*
	for(i = 0; i < rank_ + 1; i++){
	for(j = 0; j < rank_ + 1; j++)  printf("%20f", a[i][j]);
	printf("%20f\n", b[i]);
	}
	printf("\n");
	*/
	b[rank_ + 1 - 1] /= a[rank_ + 1 - 1][rank_ + 1 - 1];
	for (i = rank_ + 1 - 2; i >= 0; i--) {
		float sum = 0;
		for (j = i + 1; j < rank_ + 1; j++)  sum += a[i][j] * b[j];
		b[i] = (b[i] - sum) / a[i][i];
	}


	/*printf("P(x) = %f + %f x + %f x^2\n", b[0], b[1], b[2]);*/
	float detx = -b[1] / (2 * b[2]);
	return detx;


}

void getCentroid(float *p_img, int arraySize, int n_stream, float *xc, float *yc)
{
	int LEN = arraySize*arraySize;
	float temp = 0;
	float A = 0;
	float *array_med3 = (float*)malloc(n_stream * sizeof(float));
	getMean(p_img, array_med3, LEN, n_stream);
	for (int i = 0; i < n_stream; i++)
	{
		int i_offset = i*LEN;
		temp = 0;
		A = 0;
		xc[i] = 0;
		yc[i] = 0;
		for (int j = i_offset; j < LEN + i_offset; j++)
		{
			int arr_x = (j - i_offset) % arraySize;
			int arr_y = (j - i_offset) / arraySize;
			float temp = p_img[j] - array_med3[i];
			temp = fabs(temp);
			A += temp;
			xc[i] += (temp)*arr_x;
			yc[i] += (temp)*arr_y;
		}
		xc[i] = xc[i] / A;
		yc[i] = yc[i] / A;
		//printf("the center of array is %f, %f\n", xc[i], yc[i]);

	}


	free(array_med3);
	array_med3 = NULL;
}

void getCentroid_MN(float *p_img, int arraySize, int n_frame, int n_stream, float *xc, float *yc)
{
	int LEN = arraySize*arraySize;
	int s_LEN = LEN*n_frame;
	float temp = 0;
	float A = 0;
	float *array_med3 = (float*)calloc(n_stream*n_frame, sizeof(float));  
	getMean_MN(p_img, array_med3, LEN, n_frame, n_stream);

	for (int k = 0; k<n_stream; k++)
	{
		int s_offset = k*n_frame*LEN;
		for (int i = 0; i < n_frame; i++)
		{
			int i_offset = i*LEN;
			temp = 0;
			A = 0;
			xc[i + k*n_frame] = 0;
			yc[i + k*n_frame] = 0;
			for (int j = 0; j < LEN; j++)
			{
				int arr_x = j % arraySize;
				int arr_y = j / arraySize;
				float temp = p_img[j + i_offset + s_offset] - array_med3[i + k*n_frame];
				temp = fabs(temp);
				A += temp;
				xc[i + k*n_frame] += (temp)*arr_x;
				yc[i + k*n_frame] += (temp)*arr_y;
			}
			xc[i + k*n_frame] = xc[i + k*n_frame] / A;
			yc[i + k*n_frame] = yc[i + k*n_frame] / A;
			//printf("the center of array is %f, %f\n", xc[i+k*n_frame], yc[i+ k*n_frame]);

		}
	}

	free(array_med3);
	//*array_med3 = NULL;


}




bool cuda_QI4(float img_mat_4d[][5][50][50], int arraySize, int n_frame, int n_stream, float rdp_array[][5][57], float xc_array[][5], float yc_array[][5],float rdp_mean[][57])
{

	size_t img_bytes = arraySize*arraySize * sizeof(float);   
	size_t f_bytes = sizeof(float);

	//hipStream_t *streams = (hipStream_t *)malloc(n_stream * sizeof(hipStream_t));

	int LEN = arraySize*arraySize;                            
	int s_LEN = LEN*n_frame;


	float *p_img_4d = &img_mat_4d[0][0][0][0];
	float *rdp_profile = &rdp_array[0][0][0];
	float *xc = &xc_array[0][0];
	float *yc = &yc_array[0][0];

	//getCentroid_MN(p_img_4d, arraySize, n_frame, n_stream, xc, yc);

	float r_step = 0.4;
	int theta_num_perQ = 4;
	int t_size = theta_num_perQ * 4;
	int r_max = arraySize / 2 - 2;
	int r_N = r_max / r_step;


	size_t r_bytes = r_N * sizeof(float);
	size_t t_bytes = t_size * sizeof(float);
	float *r_sample = (float *)malloc(r_bytes);
	if (*r_sample = NULL)
		printf("out of memory!");

	float *t_sample = (float *)malloc(t_bytes);
	if (*t_sample = NULL)
		printf("out of memory!");
	//printf("good now!");



	for (int i = 0; i < r_N; i++)
	{
		r_sample[i] = i*r_step;
		//printf("the r_sequence is :%f\n", r_sample[i]);
	}

	for (int i = 0; i < t_size; i++)
	{
		t_sample[i] = i*PI * 2 / t_size;
		//printf("the t_sequence is :%f\n", t_sample[i]);
	}



	size_t rdp_bytes = r_N*t_size * sizeof(float);




	float * radial_profile = (float *)malloc(rdp_bytes*n_frame*n_stream);  

																		   // GPU allocation
	float *d_radial_profile = NULL;
	float *d_r_sample = NULL;
	float *d_t_sample = NULL;
	float *d_img_mat = NULL;
	float *d_xc = NULL;
	float *d_yc = NULL;
	////

	CHECK(hipHostMalloc(&d_radial_profile, rdp_bytes*n_frame*n_stream));
	CHECK(hipHostMalloc(&d_r_sample, r_bytes));
	CHECK(hipHostMalloc(&d_t_sample, t_bytes));
	CHECK(hipHostMalloc(&d_img_mat, img_bytes*n_frame*n_stream));
	CHECK(hipHostMalloc(&d_xc, sizeof(float)*n_frame*n_stream));
	CHECK(hipHostMalloc(&d_yc, sizeof(float)*n_frame*n_stream));
	//CHECK(hipHostMalloc(&d_check, rdp_bytes*n_frame*n_stream));

	hipStream_t *streams = (hipStream_t *)malloc(n_stream * sizeof(hipStream_t));

	for (int i = 0; i < n_stream; i++)
	{
		CHECK(hipStreamCreate(&streams[i]));
	}


	for (int i = 0; i < n_stream; i++)
	{
		int s_offset = i*n_frame*LEN;

		//for (int j = 0; j < n_frame; j++)
		//{
		//	int offset = j*LEN;
		//	int r_offset = j*r_N;
		//	int t_offset = j*t_size;

		CHECK(hipMemcpyAsync(d_r_sample, r_sample, r_bytes, hipMemcpyHostToDevice, streams[i]));
		CHECK(hipMemcpyAsync(d_t_sample, t_sample, t_bytes, hipMemcpyHostToDevice, streams[i]));
		CHECK(hipMemcpyAsync(d_img_mat + s_offset, p_img_4d + s_offset, img_bytes*n_frame, hipMemcpyHostToDevice, streams[i]));
	}



	//for (int i = 0; i < n_stream; i++)
	//{
	//	CHECK(hipStreamSynchronize(streams[i]));
	//}
	int s_rdp_LEN = r_N*t_size*n_frame;
	const dim3 block2(BLK_DIM);
	const dim3 grid2((s_rdp_LEN + block2.x - 1) / block2.x, 1);


	//printf("grid.x %d, grid.y %d,grid.z,%d\n",grid2.x,grid2.y,grid2.z);
	//printf("block.x %d, block.y %d,block.z,%d\n", block2.x, block2.y, block2.z);

	size_t rdpxy_byte = (r_N*theta_num_perQ * 4 * 2 - 1) * sizeof(float);
	float *rdp_x = (float *)malloc(rdpxy_byte);
	float *rdp_y = (float *)malloc(rdpxy_byte);


	float *rdp_x_rev = (float *)malloc(rdpxy_byte);
	float *rdp_y_rev = (float *)malloc(rdpxy_byte);


	size_t corr_bytes = (r_N*theta_num_perQ * 4 * 4 - 3) * sizeof(float);
	float *rdp_x_corr = (float*)malloc(corr_bytes);
	float *rdp_y_corr = (float*)malloc(corr_bytes);

	float *p_rdp_mean = (float *)calloc(r_N*n_stream , sizeof(float));





	for (int j = 0; j < 3; j++)
	{
		//for (int i = 0; i < n_stream; i++)
		//{
		//	int s_xy_offset = i*n_frame;
		CHECK(hipMemcpyAsync(d_xc, xc, sizeof(float)*n_frame*n_stream, hipMemcpyHostToDevice, streams[0]));
		CHECK(hipMemcpyAsync(d_yc, yc, sizeof(float)*n_frame*n_stream, hipMemcpyHostToDevice, streams[0]));
		//}

		for (int i = 0; i < n_stream; i++)
		{
			int s_img_offset = i*n_frame*LEN;
			int s_xy_offset = i*n_frame;
			int s_r_offset = i*n_frame*r_N;
			int s_t_offset = i*n_frame*t_size;
			int s_rdp_offset = i*n_frame*r_N*t_size;
			compute_radial_profile_MN << <grid2, block2, 0, streams[i] >> >
				(d_img_mat + s_img_offset, n_frame, n_stream, d_xc + s_xy_offset, d_yc + s_xy_offset, d_r_sample, d_t_sample, t_size, r_N, d_radial_profile + s_rdp_offset, arraySize);
		}



		for (int s = 0; s < n_stream; s++)
		{

			int s_rdp_offset = s*n_frame*r_N*t_size;
			int s_xy_offset = s*n_frame;
			CHECK(hipMemcpyAsync(radial_profile + s_rdp_offset, d_radial_profile + s_rdp_offset, rdp_bytes*n_frame, hipMemcpyDeviceToHost, streams[s]));


			//CHECK(hipMemcpyAsync(check + s_rdp_offset, d_check + s_rdp_offset, rdp_bytes*n_frame, hipMemcpyDeviceToHost, streams[s]));

			//FILE *fr = fopen("rdp.txt", "w");
			//if (fr == NULL)
			//{
			//	exit(1);
			//}

			//for (int t = 0; t < r_N*n_frame; t++)
			//{
			//	for (int t2 = 0; t2 < t_size; t2++)
			//	{
			//	fprintf(fr, "%f\t ",check[t2+t*t_size]);
			//	}
			//	fprintf(fr, "%\n");
			//}
			//fclose(fr);

			int s_rN_offset = s*n_frame*r_N;

			for (int i = 0; i < n_frame; i++)
			{
				int rdp_offset = i*r_N*t_size;
				int rN_offset = i*r_N;

				getrdp_x_y(radial_profile + rdp_offset + s_rdp_offset, r_N, theta_num_perQ, rdp_profile + rN_offset + s_rN_offset, rdp_x, rdp_y);

				get_rev_rdp(rdp_x, rdp_x_rev, 2 * r_N - 1);

				get_rev_rdp(rdp_y, rdp_y_rev, 2 * r_N - 1);



				rdp_corr(rdp_x, rdp_x_rev, rdp_x_corr, r_N * 2 - 1);
				rdp_corr(rdp_y, rdp_y_rev, rdp_y_corr, r_N * 2 - 1);


				int pkx, pky;
				pkx = findpeak(rdp_x_corr, r_N * 4 - 3);
				pky = findpeak(rdp_y_corr, r_N * 4 - 3);

				//FILE *fr2 = fopen("rdp_corr_x.txt", "w");
				//if (fr2 == NULL)
				//{
				//	exit(1);
				//}
				//for (int t2 = 0; t2 < (r_N * 4 - 3); t2++)
				//{
				//	fprintf(fr, "%f\n", rdp_x_corr[t2]);
				//}
				//fclose(fr2);

				float pkx_value[5] = { 0 };
				float pkx_index[5] = { 0 };
				float pky_value[5] = { 0 };
				float pky_index[5] = { 0 };
				for (int k = 0; k < 5; k++)
				{
					pkx_value[k] = rdp_x_corr[pkx - 2 + k];
					pky_value[k] = rdp_y_corr[pky - 2 + k];
					pkx_index[k] = (-(2 * r_N - 1) + (pkx - 2 + k))*r_step;
					pky_index[k] = (-(2 * r_N - 1) + (pky - 2 + k))*r_step;
				}


				float detx = LeastSquareGuassian(pkx_index, pkx_value, 5);
				float dety = LeastSquareGuassian(pky_index, pky_value, 5);

				//printf("the deviation of center is %f, %f\n", detx, dety);

				xc[i + s_xy_offset] = xc[i + s_xy_offset] - 2 * detx / PI;
				yc[i + s_xy_offset] = yc[i + s_xy_offset] - 2 * dety / PI;

			}
		}
	}






	//printf("good until now!\n");

	for (int i = 0; i < n_stream; i++)
	{
		CHECK(hipStreamDestroy(streams[i]));
	}

	//for (int i = 0; i < n_stream; i++)
	//{
	//	int xy_offset = i*n_frame;
	//	for (int j = 0; j < n_frame; j++)
	//	{
	//		printf("the real center of image is %f, %f\n", xc[j+xy_offset], yc[j+xy_offset]);
	//	}

	//}

  for (int i = 0; i < n_stream; i++)
{
  int s_mean_offset = i*r_N;
  int s_rN_offset = i*n_frame*r_N;
  for (int j = 0; j < n_frame; j++)
  {
    int rN_offset = j*r_N;
    for (int k = 0; k < r_N; k++)
    {
      p_rdp_mean[k + s_mean_offset] = p_rdp_mean[k + s_mean_offset] + rdp_profile[k + rN_offset + s_rN_offset] / n_frame;
    }
  }

}

	memcpy(rdp_mean, p_rdp_mean, r_N*n_stream * sizeof(float));



	free(r_sample);
	r_sample = NULL;
	free(t_sample);
	t_sample = NULL;


	free(radial_profile);
	radial_profile = NULL;
	free(rdp_x);
	rdp_x = NULL;
	free(rdp_y);
	rdp_y = NULL;

	free(p_rdp_mean);
	p_rdp_mean = NULL;
	free(rdp_x_rev);
	rdp_x_rev = NULL;
	free(rdp_y_rev);
	rdp_y_rev = NULL;
	free(rdp_x_corr);
	rdp_x_corr = NULL;
	free(rdp_y_corr);
	rdp_y_corr = NULL;




	////
	////
	CHECK(hipHostFree(d_radial_profile));
	CHECK(hipHostFree(d_r_sample));
	CHECK(hipHostFree(d_t_sample));
	CHECK(hipHostFree(d_img_mat));
	CHECK(hipHostFree(d_xc));
	CHECK(hipHostFree(d_yc));

	return true;
}
