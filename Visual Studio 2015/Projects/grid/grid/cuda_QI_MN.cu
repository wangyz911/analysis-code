#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <conio.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>  // ������������





#define PI 3.141592653

#define BLK_DIM 128

#define CHECK(call)  \
     { const hipError_t error = call;         \
       if (error != hipSuccess) {            \
            printf("Error:%s:%d, ",__FILE__,__LINE__);  \
            printf("code:%d, reason:%s\n",error,hipGetErrorString(error)); \
            exit(1);     \
          }}   

//void * alloc(size_t size)
//{
//	void *new_mem;
//	new_mem = malloc(size);
//	if (new_mem == NULL)
//	{
//		printf("Out of memory!\n");
//		exit(1);
//
//	}
//	return new_mem;
//}

/*Declare statically six arrays of ARRAY_SIZE each
** ����CPU����������Լ�������ŷ��ؽ���ľ���
//*/
//float cpu_cen_mat_x[ARRAY_SIZE_Y][ARRAY_SIZE_X];
//float cpu_cen_mat_y[ARRAY_SIZE_Y][ARRAY_SIZE_X];
//float cpu_yc[1];
//float cpu_xc[1];
//float img_mat[ARRAY_SIZE_Y][ARRAY_SIZE_X] = { 0 };  // ������ó�ʼ��һ�£� ��Ȼ���ױ���


int compInc(const void *a, const void *b);







/*���к������ڼ���ͼ������ĸ����޵�radial profile*/
__global__ void compute_radial_profile_MN
(float *img_array, int n_frame,int n_stream,float *xc, float *yc, float *r_sample, float *t_sample, int t_size, int r_size, float *radial_profile, int arraySize)
{
	const unsigned int idx = (blockIdx.x*blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y*blockDim.y) + threadIdx.y;
	int thread_idx = ((gridDim.x*blockDim.x)*idy) + idx;
	//if (thread_idx > (t_size*r_size*n_frame*n_stream))
	//	return;
	int s = thread_idx / (r_size*t_size);  //��i ���ж��ǵڼ���ͼ�����ݣ�Ȼ��ѡȡ��ͬ����������
	int img_offset = s*arraySize*arraySize;
	int inner_thread_idx = thread_idx % (r_size*t_size);
	int id_r = inner_thread_idx / t_size ;
	int id_t = inner_thread_idx % t_size ;
	float x = xc[s] + r_sample[id_r ] * cos(t_sample[id_t]);
	float y = yc[s] + r_sample[id_r] *  sin(t_sample[id_t]);
	int x1 = (int)x;
	int y1 = (int)y;
	int x2 = x1 + 1;
	int y2 = y1 + 1;
	float rdp = 
		img_array[y1*arraySize + x1+img_offset] * ((float)x2 - x)*((float)y2 - y) + \
		img_array[y2*arraySize + x1+img_offset] * ((float)x2 - x)*(y - (float)y1) + \
		img_array[y1*arraySize + x2+img_offset] * (x - (float)x1)*((float)y2 - y) + \
		img_array[y2*arraySize + x2+img_offset] * (x - (float)x1)*(y - (float)y1);
	float power_id = (float)(id_r) / r_size;

	float fall = 1.0 - exp(-(1.0 - power_id)*(1.0 - power_id) / 0.05);
	float rise = 1.0 - exp(-power_id*power_id / 0.01);
	float power = sqrt(fall*rise*power_id * 2.0);
	radial_profile[thread_idx] = rdp*power;
	//d_check[thread_idx] = thread_idx;


}





// �´���һ�������ֵ�ĺ���
void getMean_MN(float *img_mat, float *arrayMed, int len, int n_frame,int n_stream)
{
	for (int k = 0; k < n_stream; k++)
	{
		int s_offset = k*len*n_frame;
		
     	for (int j = 0; j < n_frame; j++)
	   {
			int offset = len*j;
		    arrayMed[j+k*n_frame] = 0;
		for (int i = 0; i < len; i++)
		  {
			arrayMed[j+k*n_frame] += img_mat[i+offset+s_offset];
		  }
		arrayMed[j+ k*n_frame] = arrayMed[j+k*n_frame] / len;
	}
	}
}


void getrdp_x_y(float *rdp_matrix, int r_N, int theta_num_perQ, float *rdp_all, float *rdp_x, float *rdp_y);

/*���º��������õ�radial profile �ķ�ת��Ϊ�������׼��*/
void get_rev_rdp(float *rdp, float *rdp_rev, int rdp_x_size);

/*���º������������������еĻ�������ߣ�����rdp_corr�����ս����N��rdp�Ĵ�С*/
void rdp_corr(float *rdp, float *rdp_rev, float *rdp_corr, int N);


/*���º�������Ѱ�����ߵķ�ֵ������������λ��*/
float findpeak(float *rdp_corr, int N);



float LeastSquareGuassian(float *arr_x, float *arr_y, int arr_N);



void getCentroid(float *p_img, int arraySize, int n_stream, float *xc, float *yc);


void getCentroid_MN(float *p_img, int arraySize, int n_frame,int n_stream, float *xc, float *yc)
{
	int LEN = arraySize*arraySize;
	int s_LEN = LEN*n_frame;
	float temp = 0;
	float A = 0;
	float *array_med3 = (float*)calloc(n_stream*n_frame , sizeof(float));  // �˴�ʹ���Դ���ʼ��Ϊ0 ��calloc����
	getMean_MN(p_img, array_med3, LEN,n_frame, n_stream);

	for (int k=0;k<n_stream;k++)
	{
		int s_offset = k*n_frame*LEN;
	for (int i = 0; i < n_frame; i++)
	{
		int i_offset = i*LEN;
		temp = 0;
		A = 0;
		xc[i+k*n_frame] = 0;
		yc[i+k*n_frame] = 0;
		for (int j = 0; j < LEN; j++)
		{
			int arr_x = j % arraySize;
			int arr_y = j / arraySize;
			float temp = p_img[j+i_offset+s_offset] - array_med3[i+k*n_frame];
			temp = fabs(temp);
			A += temp;
			xc[i+ k*n_frame] += (temp)*arr_x;
			yc[i+ k*n_frame] += (temp)*arr_y;
		}
		xc[i+ k*n_frame] = xc[i+ k*n_frame] / A;
		yc[i+ k*n_frame] = yc[i+ k*n_frame] / A;
		//printf("the center of array is %f, %f\n", xc[i+k*n_frame], yc[i+ k*n_frame]);

	}
	}

	free(array_med3);
	//*array_med3 = NULL;


}









/////////////////////////////////////////////////////////////////////////////////////////////////////
void cuda_QI4(float *p_img_4d, int arraySize, int n_frame,int n_stream, float *rdp_profile, float *xc, float *yc)
{
	//int arraySize = 80;

	size_t img_bytes = arraySize*arraySize * sizeof(float);   // ����Ԫ����ռ�ռ�
	size_t f_bytes = sizeof(float);

	//hipStream_t *streams = (hipStream_t *)malloc(n_stream * sizeof(hipStream_t));

	int LEN = arraySize*arraySize;                            // ͼ�����Ԫ����
	int s_LEN = LEN*n_frame;

	// ����getCentroid ��������n_stream ��ͼ������xc,yc

	getCentroid_MN(p_img_4d, arraySize, n_frame,n_stream, xc, yc);
	//printf("please be good!\n");


	/// ������ʵ��QI�㷨  
	// �ȶ�����صı�����
	float r_step = 0.4;
	int theta_num_perQ =4;
	int t_size = theta_num_perQ * 4;
	int r_max = arraySize / 2 - 2;
	int r_N = r_max / r_step;

	// ���������㣬��Щ��������е�ROI������һ����
	size_t r_bytes = r_N * sizeof(float);
	size_t t_bytes = t_size * sizeof(float);
	float *r_sample = (float *)malloc(r_bytes*n_frame);
	if (*r_sample = NULL)
		printf("out of memory!");

	float *t_sample = (float *)malloc(t_bytes*n_frame);
	if (*t_sample = NULL)
		printf("out of memory!");
	//printf("good now!");



for (int i = 0; i < r_N; i++)
{
	r_sample[i] = i*r_step;
	//printf("the r_sequence is :%f\n", r_sample[i]);
}

for (int i = 0; i < t_size; i++)
   {
	   t_sample[i] = i*PI * 2 / t_size;
		//printf("the t_sequence is :%f\n", t_sample[i]);
	}


	//Ԥ����QI�˺�����ؽ���������Լ���Ӧ��GPU����
	size_t rdp_bytes = r_N*t_size * sizeof(float);
	size_t rdp_s_bytes = rdp_bytes*n_frame;



	float * radial_profile = (float *)malloc(rdp_bytes*n_frame*n_stream);  // �������radial profile������
	//float * check = (float *)malloc(rdp_bytes*n_frame*n_stream);  //�������
															// GPU allocation
	float *d_radial_profile = NULL;
	//float *d_check = NULL;
	float *d_r_sample = NULL;
	float *d_t_sample = NULL;
	float *d_img_mat = NULL;
	float *d_xc = NULL;
	float *d_yc = NULL;
	////

	CHECK(hipHostMalloc(&d_radial_profile, rdp_bytes*n_frame*n_stream));
	CHECK(hipHostMalloc(&d_r_sample, r_bytes));
	CHECK(hipHostMalloc(&d_t_sample, t_bytes));
	CHECK(hipHostMalloc(&d_img_mat, img_bytes*n_frame*n_stream));
	CHECK(hipHostMalloc(&d_xc, sizeof(float)*n_frame*n_stream));
	CHECK(hipHostMalloc(&d_yc, sizeof(float)*n_frame*n_stream));
	//CHECK(hipHostMalloc(&d_check, rdp_bytes*n_frame*n_stream));

	hipStream_t *streams = (hipStream_t *)malloc(n_stream * sizeof(hipStream_t));

	for (int i = 0; i < n_stream; i++)
	{
		CHECK(hipStreamCreate(&streams[i]));
	}


	// ��������  
	//CHECK(hipMemcpy(d_radial_profile, radial_profile, rdp_bytes, hipMemcpyHostToDevice));

	for (int i = 0; i < n_stream; i++)
	{
		int s_offset = i*n_frame*LEN;

		//for (int j = 0; j < n_frame; j++)
		//{
		//	int offset = j*LEN;
		//	int r_offset = j*r_N;
		//	int t_offset = j*t_size;

			CHECK(hipMemcpyAsync(d_r_sample, r_sample, r_bytes, hipMemcpyHostToDevice, streams[i]));
			CHECK(hipMemcpyAsync(d_t_sample, t_sample, t_bytes, hipMemcpyHostToDevice, streams[i]));
			CHECK(hipMemcpyAsync(d_img_mat+ s_offset, p_img_4d+ s_offset, img_bytes*n_frame, hipMemcpyHostToDevice, streams[i]));
		}
			// ����δ����Ϊ1�δ��䣬�ٶ�������5ms��������һ�����������������޷���ȡ����
	//}



	//for (int i = 0; i < n_stream; i++)
	//{
	//	CHECK(hipStreamSynchronize(streams[i]));
	//}
	int s_rdp_LEN = r_N*t_size*n_frame;
	const dim3 block2(BLK_DIM);
	const dim3 grid2((s_rdp_LEN + block2.x - 1) / block2.x, 1);

	//printf("grid.x %d, grid.y %d,grid.z,%d\n",grid2.x,grid2.y,grid2.z);
	//printf("block.x %d, block.y %d,block.z,%d\n", block2.x, block2.y, block2.z);



	size_t rdpxy_byte = (r_N*theta_num_perQ * 4 * 2 - 1) * sizeof(float);
	float *rdp_x = (float *)malloc(rdpxy_byte);
	float *rdp_y = (float *)malloc(rdpxy_byte);

	// ������ȡrdp_x �ķ����������η�������
	float *rdp_x_rev = (float *)malloc(rdpxy_byte);
	float *rdp_y_rev = (float *)malloc(rdpxy_byte);

	// ���������л���ؼ��㣬
	size_t corr_bytes = (r_N*theta_num_perQ * 4 * 4 - 3) * sizeof(float);
	float *rdp_x_corr = (float*)malloc(corr_bytes);
	float *rdp_y_corr = (float*)malloc(corr_bytes);

	float *p_rdp_mean = (float *)calloc(r_N*n_stream, sizeof(float));

	for (int j = 0; j < 3; j++)
	{
		//for (int i = 0; i < n_stream; i++)
		//{
		//	int s_xy_offset = i*n_frame;
			CHECK(hipMemcpyAsync(d_xc, xc, sizeof(float)*n_frame*n_stream, hipMemcpyHostToDevice,streams[0]));
			CHECK(hipMemcpyAsync(d_yc, yc, sizeof(float)*n_frame*n_stream, hipMemcpyHostToDevice,streams[0]));
		//}
		// ���������һ�£��Ѵ���XY�ͼ�����һ��ѭ����ִ���ٶȲ��䣬������������΢�ӳ٣����ռӳ��ܵ�ִ��ʱ��

			// ����һ�·ֽ�������������GPU��CPU��ִ��ʱ��
			// ��һ������һ��GPU����
		for (int i = 0; i < n_stream; i++)
		{
			int s_img_offset = i*n_frame*LEN;
			int s_xy_offset = i*n_frame;
			int s_r_offset = i*n_frame*r_N;
			int s_t_offset = i*n_frame*t_size;
			int s_rdp_offset = i*n_frame*r_N*t_size;
			compute_radial_profile_MN << <grid2, block2, 0, streams[i] >> >
				(d_img_mat+s_img_offset, n_frame,n_stream , d_xc+s_xy_offset,d_yc + s_xy_offset, d_r_sample, d_t_sample, t_size, r_N, d_radial_profile+ s_rdp_offset, arraySize);
		}

		// �ϲ�������������ѭ����GPU����ʱ��ֱ������һ����û�в��п���

		for (int s = 0; s < n_stream; s++)
		{

				int s_rdp_offset = s*n_frame*r_N*t_size;
				int s_xy_offset = s*n_frame;
				CHECK(hipMemcpyAsync(radial_profile + s_rdp_offset, d_radial_profile + s_rdp_offset, rdp_bytes*n_frame, hipMemcpyDeviceToHost, streams[s]));
				//CHECK(hipMemcpyAsync(check + s_rdp_offset, d_check + s_rdp_offset, rdp_bytes*n_frame, hipMemcpyDeviceToHost, streams[s]));

		//FILE *fr = fopen("rdp.txt", "w");
		//if (fr == NULL)
		//{
		//	exit(1);
		//}

		//for (int t = 0; t < r_N*n_frame; t++)
		//{
		//	for (int t2 = 0; t2 < t_size; t2++)
		//	{
		//	fprintf(fr, "%f\t ",check[t2+t*t_size]);
		//	}
		//	fprintf(fr, "%\n");
		//}
		//fclose(fr);

			int s_rN_offset = s*n_frame*r_N;
		
		for (int i = 0; i < n_frame; i++)
		{
			int xy_offset = i;
			int rdp_offset = i*r_N*t_size;
			int rN_offset = i*r_N;
			// radial profile ��X Y�����Ϻϲ�
			getrdp_x_y(radial_profile + rdp_offset + s_rdp_offset, r_N, theta_num_perQ, rdp_profile + rN_offset+s_rN_offset, rdp_x, rdp_y);






			get_rev_rdp(rdp_x, rdp_x_rev, 2 * r_N - 1);

			get_rev_rdp(rdp_y, rdp_y_rev, 2 * r_N - 1);



			rdp_corr(rdp_x, rdp_x_rev, rdp_x_corr, r_N * 2 - 1);
			rdp_corr(rdp_y, rdp_y_rev, rdp_y_corr, r_N * 2 - 1);

			// Ѱ�����ֵ������ֵ����������
			int pkx, pky;
			pkx = findpeak(rdp_x_corr, r_N * 4 - 3);
			pky = findpeak(rdp_y_corr, r_N * 4 - 3);

			//FILE *fr2 = fopen("rdp_corr_x.txt", "w");
			//if (fr2 == NULL)
			//{
			//	exit(1);
			//}
			//for (int t2 = 0; t2 < (r_N * 4 - 3); t2++)
			//{
			//	fprintf(fr, "%f\n", rdp_x_corr[t2]);
			//}
			//fclose(fr2);
			// ��ȡ���ֵ������5���㣬���������±�
			float pkx_value[5] = { 0 };
			float pkx_index[5] = { 0 };
			float pky_value[5] = { 0 };
			float pky_index[5] = { 0 };
			for (int k = 0; k < 5; k++)
			{
				pkx_value[k] = rdp_x_corr[pkx - 2 + k];
				pky_value[k] = rdp_y_corr[pky - 2 + k];
				pkx_index[k] = (-(2 * r_N - 2) + (pkx - 2 + k))*r_step;
				pky_index[k] = (-(2 * r_N - 2) + (pky - 2 + k))*r_step;
			}

			// �������ֱ�����x y�����������߼�ֵ�㣬��������С���˷��������������

			float detx = LeastSquareGuassian(pkx_index, pkx_value, 5);
			float dety = LeastSquareGuassian(pky_index, pky_value, 5);

			//printf("the deviation of center is %f, %f\n", detx, dety);

			xc[i+ s_xy_offset] = xc[i+ s_xy_offset] - 2 * detx / PI;
			yc[i+ s_xy_offset] = yc[i+ s_xy_offset]- 2 * dety / PI;

		}
		}
	}

	// �����Ѿ������һ�ִ�ͼ��������������ȫ�����̣�ʣ�µľ��ǵ���QI��
	//printf("good until now!\n");

	for (int i = 0; i < n_stream; i++)
	{
		CHECK(hipStreamDestroy(streams[i]));
	}

	for (int i = 0; i < n_stream; i++)
	{
		int s_mean_offset = i*r_N;
		int s_rN_offset = i*n_frame*r_N;
		for (int j = 0; j < n_frame; j++)
		{
			int rN_offset = j*r_N;
			for (int k = 0; k < r_N; k++)
			{
				p_rdp_mean[k + s_mean_offset] = p_rdp_mean[k + s_mean_offset] + rdp_profile[k + rN_offset + s_rN_offset] / n_frame;
			}
		}

	}





	//for (int i = 0; i < n_stream; i++)
	//{
	//	for (int j = 0; j < r_N; j++)
	//	{
	//		printf("the rdp_mean is %f\n", p_rdp_mean[j + i*r_N]);
	//	}
	//}
	//for (int i = 0; i < n_stream; i++)
	//{
	//	int xy_offset = i*n_frame;
	//	for (int j = 0; j < n_frame; j++)
	//	{
	//		printf("the real center of image is %f, %f\n", xc[j+xy_offset], yc[j+xy_offset]);
	//	}

	//}


	/////// �ͷ��ڴ�ռ�

	free(r_sample);
	r_sample = NULL;
	free(t_sample);
	t_sample = NULL;
	//free(check);
	//check = NULL;



	free(radial_profile);
	radial_profile = NULL;
	free(rdp_x);
	rdp_x = NULL;
	free(rdp_y);
	rdp_y = NULL;

	free(rdp_x_rev);
	rdp_x_rev = NULL;
	free(rdp_y_rev);
	rdp_y_rev = NULL;
	free(rdp_x_corr);
	rdp_x_corr = NULL;
	free(rdp_y_corr);
	rdp_y_corr = NULL;

	free(p_rdp_mean);
	p_rdp_mean = NULL;


	////
	////
	//CHECK(hipHostFree(d_check));
	CHECK(hipHostFree(d_radial_profile));
	CHECK(hipHostFree(d_r_sample));
	CHECK(hipHostFree(d_t_sample));
	CHECK(hipHostFree(d_img_mat));
	CHECK(hipHostFree(d_xc));
	CHECK(hipHostFree(d_yc));

}


