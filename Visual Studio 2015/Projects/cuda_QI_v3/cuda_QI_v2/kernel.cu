#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <conio.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>  // 用来复制数组
#include "gpu_func.cuh"


#define PI 3.141592653
#define BLK_DIM2 32
#define CHECK(call)  \
     { const hipError_t error = call;         \
       if (error != hipSuccess) {            \
            printf("Error:%s:%d, ",__FILE__,__LINE__);  \
            printf("code:%d, reason:%s\n",error,hipGetErrorString(error)); \
            exit(1);     \
          }}

//int compInc(const void *a, const void *b)
//{
//	return *(int *)a - *(int *)b;
//}
//
//__global__ void compute_radial_profile
//(float *img_array, float xc, float yc, float *r_sample, float *t_sample, int t_size, int r_size, float *radial_profile, int arraySize)
//{
//	const unsigned int idx = (blockIdx.x*blockDim.x) + threadIdx.x;
//	const unsigned int idy = (blockIdx.y*blockDim.y) + threadIdx.y;
//	int thread_idx = ((gridDim.x*blockDim.x)*idy) + idx;
//	//int i = thread_idx / (arraySize*arraySize); 
//
//	//int inner_thread_idx = thread_idx % (arraySize*arraySize);
//	int id_r = thread_idx / t_size;
//	int id_t = thread_idx % t_size;
//	float x = xc + r_sample[id_r] * sin(t_sample[id_t]);
//	float y = yc + r_sample[id_r] * cos(t_sample[id_t]);
//	int x1 = (int)x;
//	int y1 = (int)y;
//	int x2 = x1 + 1;
//	int y2 = y1 + 1;
//	float rdp = img_array[y1*arraySize + x1] * ((float)x2 - x)*((float)y2 - y) + \
//		img_array[y2*arraySize + x1] * ((float)x2 - x)*(y - (float)y1) + \
//		img_array[y1*arraySize + x2] * (x - (float)x1)*((float)y2 - y) + \
//		img_array[y2*arraySize + x2] * (x - (float)x1)*(y - (float)y1);
//	float power_id = (float)(id_r) / r_size;
//
//	float fall = 1.0 - exp(-(1.0 - power_id)*(1.0 - power_id) / 0.05);
//	float rise = 1.0 - exp(-power_id*power_id / 0.01);
//	float power = sqrt(fall*rise*power_id * 2.0);
//	radial_profile[thread_idx] = rdp*power;
//
//}
//
///*下列函数用于计算图像矩阵四个象限的radial profile*/
//__global__ void compute_radial_profile_MN
//(float *img_array, float *xc, float *yc, float *r_sample, float *t_sample, int t_size, int r_size, float *radial_profile, int arraySize)
//{
//	const unsigned int idx = (blockIdx.x*blockDim.x) + threadIdx.x;
//	const unsigned int idy = (blockIdx.y*blockDim.y) + threadIdx.y;
//	int thread_idx = ((gridDim.x*blockDim.x)*idy) + idx;
//	int s = thread_idx / (r_size*t_size);  //用i 来判断是第几张图的数据，然后选取不同的质心坐标
//	int img_offset = s*arraySize*arraySize;
//	int r_offset = s*r_size;
//	int t_offset = s*t_size;
//	int inner_thread_idx = thread_idx % (r_size*t_size);
//	int id_r = inner_thread_idx / t_size;
//	int id_t = inner_thread_idx % t_size;
//	float x = xc[s] + r_sample[id_r + r_offset] * sin(t_sample[id_t + t_offset]);
//	float y = yc[s] + r_sample[id_r + r_offset] * cos(t_sample[id_t + t_offset]);
//	int x1 = (int)x;
//	int y1 = (int)y;
//	int x2 = x1 + 1;
//	int y2 = y1 + 1;
//	float rdp =
//		img_array[y1*arraySize + x1 + img_offset] * ((float)x2 - x)*((float)y2 - y) + \
//		img_array[y2*arraySize + x1 + img_offset] * ((float)x2 - x)*(y - (float)y1) + \
//		img_array[y1*arraySize + x2 + img_offset] * (x - (float)x1)*((float)y2 - y) + \
//		img_array[y2*arraySize + x2 + img_offset] * (x - (float)x1)*(y - (float)y1);
//	float power_id = (float)(id_r) / r_size;
//
//	float fall = 1.0 - exp(-(1.0 - power_id)*(1.0 - power_id) / 0.05);
//	float rise = 1.0 - exp(-power_id*power_id / 0.01);
//	float power = sqrt(fall*rise*power_id * 2.0);
//	radial_profile[thread_idx] = rdp*power;
//
//
//}
//
//
//
//void getMean(float *img_mat, float *arrayMed, int len, int n_stream)
//{
//	for (int j = 0; j < n_stream; j++)
//	{
//		arrayMed[j] = 0;
//		for (int i = j*len; i < (j + 1)*len; i++)
//		{
//			arrayMed[j] += img_mat[i];
//		}
//		arrayMed[j] = arrayMed[j] / len;
//	}
//
//}
//
//void getMean_MN(float *img_mat, float *arrayMed, int len, int n_frame, int n_stream)
//{
//	for (int k = 0; k < n_stream; k++)
//	{
//		int s_offset = k*len*n_frame;
//
//		for (int j = 0; j < n_frame; j++)
//		{
//			int offset = len*j;
//			arrayMed[j + k*n_frame] = 0;
//			for (int i = 0; i < len; i++)
//			{
//				arrayMed[j + k*n_frame] += img_mat[i + offset + s_offset];
//			}
//			arrayMed[j + k*n_frame] = arrayMed[j + k*n_frame] / len;
//		}
//	}
//}
///* 以下函数根据radial profile 矩阵直接计算x方向上的radial profile 和Y 方向上的radial profile */
//
//void getrdp_x_y(float *rdp_matrix, int r_N, int theta_num_perQ, float *rdp_all, float *rdp_x, float *rdp_y)
//{
//	size_t r_byte = sizeof(float)*r_N;
//	float *rdp_1 = (float *)malloc(r_byte);
//	float *rdp_2 = (float *)malloc(r_byte);
//	float *rdp_3 = (float *)malloc(r_byte);
//	float *rdp_4 = (float *)malloc(r_byte);
//
//	for (int i = 0; i < r_N; i++)
//	{
//		rdp_1[i] = 0;
//		rdp_2[i] = 0;
//		rdp_3[i] = 0;
//		rdp_4[i] = 0;
//
//		for (int j = 0; j < theta_num_perQ; j++)
//		{
//			rdp_1[i] = rdp_1[i] + rdp_matrix[j + 4 * i*theta_num_perQ];
//
//			rdp_2[i] = rdp_2[i] + rdp_matrix[j + 1 * theta_num_perQ + 4 * i*theta_num_perQ];
//
//			rdp_3[i] = rdp_3[i] + rdp_matrix[j + 2 * theta_num_perQ + 4 * i*theta_num_perQ];
//
//			rdp_4[i] = rdp_4[i] + rdp_matrix[j + 3 * theta_num_perQ + 4 * i*theta_num_perQ];
//		};
//		rdp_all[i] = (rdp_1[i] + rdp_2[i] + rdp_3[i] + rdp_4[i]) / (200 * theta_num_perQ);
//	};
//
//	for (int k = 0; k < (2 * r_N - 1); k++)
//	{
//		if (k < r_N - 1)
//		{
//			rdp_x[k] = (rdp_2[r_N - 1 - k] + rdp_3[r_N - 1 - k]) / 100;
//			rdp_y[k] = (rdp_3[r_N - 1 - k] + rdp_4[r_N - 1 - k]) / 100;
//		}
//		else
//		{
//			rdp_x[k] = (rdp_1[k + 1 - r_N] + rdp_4[k + 1 - r_N]) / 100;
//			rdp_y[k] = (rdp_1[k + 1 - r_N] + rdp_2[k + 1 - r_N]) / 100;
//		}
//	}
//	free(rdp_1);
//	rdp_1 = NULL;
//	free(rdp_2);
//	rdp_2 = NULL;
//	free(rdp_3);
//	rdp_3 = NULL;
//	free(rdp_4);
//	rdp_4 = NULL;
//
//
//}
//
//void get_rev_rdp(float *rdp, float *rdp_rev, int rdp_x_size)
//{
//	for (int i = 0; i < rdp_x_size; i++)
//	{
//		rdp_rev[i] = rdp[rdp_x_size - 1 - i];
//	}
//	return;
//}
//
//void rdp_corr(float *rdp, float *rdp_rev, float *rdp_corr, int N)
//{
//	float corr_ij;
//	int    delay, i, j;
//
//	for (delay = -N + 1; delay < N; delay++)
//	{
//		//Calculate the numerator
//		corr_ij = 0;
//		for (i = 0; i < N; i++)
//		{
//			j = i + delay;
//			if ((j < 0) || (j >= N))  //The series are no wrapped,so the value is ignored
//				continue;
//			else
//				corr_ij += (rdp[i] * rdp_rev[j]);
//		}
//
//		//Calculate the correlation series at "delay"
//		rdp_corr[delay + N - 1] = corr_ij;
//	}
//}
//
//float findpeak(float *rdp_corr, int N)
//{
//	float max = rdp_corr[0];
//	int index = 0;
//	for (int i = 0; i < N; i++)
//	{
//		if (max <= rdp_corr[i])
//		{
//			index = i;
//			max = rdp_corr[i];
//		}
//		;
//	}
//	return index;
//}
//
//float LeastSquareGuassian(float *arr_x, float *arr_y, int arr_N)
//{
//	const int rank_ = 2;
//	float atemp[2 * (rank_ + 1)] = { 0 }, b[rank_ + 1] = { 0 }, a[rank_ + 1][rank_ + 1];
//	int i, j, k;
//
//	for (i = 0; i < arr_N; i++) {  //
//		atemp[1] += arr_x[i];
//		atemp[2] += pow(arr_x[i], 2);
//		atemp[3] += pow(arr_x[i], 3);
//		atemp[4] += pow(arr_x[i], 4);
//		//atemp[5] += pow(arr_x[i], 5);
//		//atemp[6] += pow(arr_x[i], 6);
//		b[0] += arr_y[i];
//		b[1] += arr_x[i] * arr_y[i];
//		b[2] += pow(arr_x[i], 2) * arr_y[i];
//		//b[3] += pow(arr_x[i], 3) * arr_y[i];
//	}
//
//	atemp[0] = arr_N;
//	/*
//	for(i = 0; i <= 2 * rank_; i++)  printf("atemp[%d] = %f\n", i, atemp[i]);
//	printf("\n");
//	for(i = 0; i <= rank_; i++)  printf("b[%d] = %f\n", i, b[i]);
//	printf("\n");
//	*/
//	for (i = 0; i < rank_ + 1; i++) {
//		k = i;
//		for (j = 0; j < rank_ + 1; j++)  a[i][j] = atemp[k++];
//	}
//	/*
//	for(i = 0; i < rank_ + 1; i++){
//	for(j = 0; j < rank_ + 1; j++)  printf("a[%d][%d] = %-17f  ", i, j, a[i][j]);
//	printf("\n");
//	}
//	printf("\n");
//	*/
//
//
//	for (k = 0; k < rank_ + 1 - 1; k++) {
//		int column = k;
//		float mainelement = a[k][k];
//
//		for (i = k; i < rank_ + 1; i++)
//			if (fabs(a[i][k]) > mainelement) {
//				mainelement = fabs(a[i][k]);
//				column = i;
//			}
//		for (j = k; j < rank_ + 1; j++)
//		{  
//			float atemp = a[k][j];
//			a[k][j] = a[column][j];
//			a[column][j] = atemp;
//		}
//		float btemp = b[k];
//		b[k] = b[column];
//		b[column] = btemp;
//
//		for (i = k + 1; i < rank_ + 1; i++) {
//			float Mik = a[i][k] / a[k][k];
//			for (j = k; j < rank_ + 1; j++)  a[i][j] -= Mik * a[k][j];
//			b[i] -= Mik * b[k];
//		}
//	}
//	/*
//	for(i = 0; i < rank_ + 1; i++){
//	for(j = 0; j < rank_ + 1; j++)  printf("%20f", a[i][j]);
//	printf("%20f\n", b[i]);
//	}
//	printf("\n");
//	*/
//	b[rank_ + 1 - 1] /= a[rank_ + 1 - 1][rank_ + 1 - 1];
//	for (i = rank_ + 1 - 2; i >= 0; i--) {
//		float sum = 0;
//		for (j = i + 1; j < rank_ + 1; j++)  sum += a[i][j] * b[j];
//		b[i] = (b[i] - sum) / a[i][i];
//	}
//
//
//	/*printf("P(x) = %f + %f x + %f x^2\n", b[0], b[1], b[2]);*/
//	float detx = -b[1] / (2 * b[2]);
//	return detx;
//
//
//}
//
//void getCentroid(float *p_img, int arraySize, int n_stream, float *xc, float *yc)
//{
//	int LEN = arraySize*arraySize;
//	float temp = 0;
//	float A = 0;
//	float *array_med3 = (float*)malloc(n_stream * sizeof(float));
//	getMean(p_img, array_med3, LEN, n_stream);
//	for (int i = 0; i < n_stream; i++)
//	{
//		int i_offset = i*LEN;
//		temp = 0;
//		A = 0;
//		xc[i] = 0;
//		yc[i] = 0;
//		for (int j = i_offset; j < LEN + i_offset; j++)
//		{
//			int arr_x = (j - i_offset) % arraySize;
//			int arr_y = (j - i_offset) / arraySize;
//			float temp = p_img[j] - array_med3[i];
//			temp = fabs(temp);
//			A += temp;
//			xc[i] += (temp)*arr_x;
//			yc[i] += (temp)*arr_y;
//		}
//		xc[i] = xc[i] / A;
//		yc[i] = yc[i] / A;
//		//printf("the center of array is %f, %f\n", xc[i], yc[i]);
//
//	}
//
//
//	free(array_med3);
//	array_med3 = NULL;
//}
//
//void getCentroid_MN(float *p_img, int arraySize, int n_frame, int n_stream, float *xc, float *yc)
//{
//	int LEN = arraySize*arraySize;
//	int s_LEN = LEN*n_frame;
//	float temp = 0;
//	float A = 0;
//	float *array_med3 = (float*)calloc(n_stream*n_frame, sizeof(float));  // 此处使用自带初始化为0 的calloc函数
//	getMean_MN(p_img, array_med3, LEN, n_frame, n_stream);
//
//	for (int k = 0; k<n_stream; k++)
//	{
//		int s_offset = k*n_frame*LEN;
//		for (int i = 0; i < n_frame; i++)
//		{
//			int i_offset = i*LEN;
//			temp = 0;
//			A = 0;
//			xc[i + k*n_frame] = 0;
//			yc[i + k*n_frame] = 0;
//			for (int j = 0; j < LEN; j++)
//			{
//				int arr_x = j % arraySize;
//				int arr_y = j / arraySize;
//				float temp = p_img[j + i_offset + s_offset] - array_med3[i + k*n_frame];
//				temp = fabs(temp);
//				A += temp;
//				xc[i + k*n_frame] += (temp)*arr_x;
//				yc[i + k*n_frame] += (temp)*arr_y;
//			}
//			xc[i + k*n_frame] = xc[i + k*n_frame] / A;
//			yc[i + k*n_frame] = yc[i + k*n_frame] / A;
//			//printf("the center of array is %f, %f\n", xc[i+k*n_frame], yc[i+ k*n_frame]);
//
//		}
//	}
//
//	free(array_med3);
//	//*array_med3 = NULL;
//
//
//}

// Texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex;

__global__ void compute_radial_profile_tex
(const float *d_xc, const float *d_yc,const float *d_x_sample, const float *d_y_sample,const float *d_power,const int t_size,const int r_size, float *d_radial_profile, const int arraySize,const int s_img_offset)
{
	const unsigned int idx = (blockIdx.x*blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y*blockDim.y) + threadIdx.y;
	int thread_idx = ((gridDim.x*blockDim.x)*idy) + idx;
	int s = thread_idx / (r_size*t_size);  //用i 来判断是第几张图的数据，然后选取不同的质心坐标 这里不应该加1
	int img_offset = s*arraySize*arraySize;
	int inner_thread_idx = thread_idx % (r_size*t_size);
	int p_id = inner_thread_idx / t_size;
	float x = d_xc[s] + d_x_sample[inner_thread_idx];
	float y = d_yc[s] + d_y_sample[inner_thread_idx] + s*arraySize+s_img_offset;
	float p = d_power[p_id];
	d_radial_profile[thread_idx] = tex2D(tex, x + 0.5f, y + 0.5f)*p;
	//d_check[thread_idx] = d_radial_profile[thread_idx];
}






bool cuda_QI4(float img_mat_4d[][5][50][50], int arraySize, int n_frame, int n_stream, float rdp_array[][5][50], float xc_array[][5], float yc_array[][5], float rdp_mean[][50])
{

	size_t img_bytes = arraySize*arraySize * sizeof(float);   // 矩阵元素所占空间
	//size_t f_bytes = sizeof(float);

	//hipStream_t *streams = (hipStream_t *)malloc(n_stream * sizeof(hipStream_t));

	//int LEN = arraySize*arraySize;                            // 图像矩阵元素数
	//int s_LEN = LEN*n_frame;

	// 调用getCentroid 函数计算n_stream 张图的质心xc,yc
	float *p_img_4d = &img_mat_4d[0][0][0][0];
	float *rdp_profile = &rdp_array[0][0][0];
	float *xc = &xc_array[0][0];
	float *yc = &yc_array[0][0];
	float *p_rdp_mean = &rdp_mean[0][0];
	//getCentroid_MN(p_img_4d, arraySize, n_frame, n_stream, xc, yc);

	/// 接下来实现QI算法  
	// 先定义相关的变量；
	float r_step = 0.4;
	int thetaPerQ = 4;
	int t_size = thetaPerQ * 4;
	//int t_size =4 ;
	int r_max = arraySize / 2 - 5;
	int r_N = r_max / r_step;

	// 创建采样点，这些点对于所有的ROI矩阵是一样的
	size_t r_bytes = r_N * sizeof(float);
	//size_t t_bytes = t_size * sizeof(float);



	//预设置QI核函数相关结果变量，以及对应的GPU部分
	size_t rdp_bytes = r_N*t_size * sizeof(float);
	size_t rdp_s_bytes = rdp_bytes*n_frame;
	/// 生成极坐标化的常数部分，加上质心坐标即可得到所有采样点坐标， 将x_sample 和y_sample 按照角度变化排列，和核函数一致，能够减少数据访问次数
	// 抛弃了生成r和t 采样点的方案，改为直接生成极坐标矩阵，r_sample 和t_sample 随算随用，覆盖重用，节省空间。
	float *x_sample = (float *)malloc(rdp_bytes);
	if (*x_sample == NULL)
		printf("out of memory!");
	float *y_sample = (float *)malloc(rdp_bytes);
	if (*y_sample == NULL)
		printf("out of memory!");
	compute_x_y(x_sample, y_sample, r_step, t_size, r_N);
	/// ***************************************************************************************************************************************                              
	/// 计算权重，并保存于常量内存
	float *power = (float *)malloc(r_bytes);

	compute_power(power, r_N);



	float * radial_profile = (float *)malloc(rdp_bytes*n_frame*n_stream);
	//float * check = (float *)malloc(rdp_bytes*n_frame*n_stream);  //检查数据
	// GPU allocation
	float *d_radial_profile = NULL;
	//float *d_check = NULL;

	float *d_img_mat = NULL;                                                            // 设备端创建图像，预备绑定纹理
	float *d_xc = NULL;
	float *d_yc = NULL;
	////
	float *d_x_sample = NULL;

	float *d_y_sample = NULL;
	float *d_power = NULL;


	CHECK(hipMalloc(&d_power, r_bytes));
	CHECK(hipMalloc(&d_x_sample, rdp_bytes));
	CHECK(hipMalloc(&d_y_sample, rdp_bytes));
	CHECK(hipHostMalloc(&d_radial_profile, rdp_bytes*n_frame*n_stream));

	//CHECK(hipMalloc(&d_img_mat, img_bytes*n_frame*n_stream));              
	CHECK(hipMalloc(&d_xc, sizeof(float)*n_frame*n_stream));
	CHECK(hipMalloc(&d_yc, sizeof(float)*n_frame*n_stream));
	//CHECK(hipHostMalloc(&d_check, rdp_bytes*n_frame*n_stream));

	hipStream_t *streams = (hipStream_t *)malloc(n_stream * sizeof(hipStream_t));

	CHECK(hipMemcpy(d_power, power, r_bytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_x_sample, x_sample, rdp_bytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_y_sample, y_sample, rdp_bytes, hipMemcpyHostToDevice));

	/// 设置纹理内存
	// Allocate array and copy image data
	size_t pitch;


	CHECK(hipMallocPitch((void**)&d_img_mat, &pitch, arraySize * sizeof(float), arraySize*n_frame*n_stream));  //// 设备端创建图像，预备绑定纹理
	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	CHECK(hipBindTexture2D(0, &tex, d_img_mat, &desc, arraySize, arraySize*n_frame*n_stream, pitch));
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.filterMode = hipFilterModeLinear;    // --- Enable linear filtering
	tex.normalized = false;                    // --- Texture coordinates will NOT be normalized
	CHECK(hipMemcpy2D(d_img_mat, pitch, p_img_4d, sizeof(float)*arraySize, sizeof(float)*arraySize, arraySize*n_frame*n_stream, hipMemcpyHostToDevice));

	// 图像矩阵已经绑定到纹理，接下来就可以在核函数中使用纹理了。

	for (int i = 0; i < n_stream; i++)
	{
		CHECK(hipStreamCreate(&streams[i]));
	}



	int s_rdp_LEN = r_N*t_size*n_frame;
	const dim3 block2(BLK_DIM2);
	const dim3 grid2((s_rdp_LEN + block2.x - 1) / block2.x, 1);



	// 设置CPU端的x y 数据（后面也许并不需要，如果直接算完了的话，但GPU端是必须设置的）
	size_t rdpxy_byte = (r_N * 2 - 1) *n_frame*n_stream * sizeof(float);
	size_t s_rdpxy_byte = (r_N * 2 - 1) *n_frame * sizeof(float);
	float *rdp_x = (float *)malloc(rdpxy_byte);
	float *rdp_y = (float *)malloc(rdpxy_byte);

	//// 接下来取rdp_x 的反，调用两次反向函数。
	//float *rdp_x_rev = (float *)malloc(rdpxy_byte);
	//float *rdp_y_rev = (float *)malloc(rdpxy_byte);

	// 接下来进行互相关计算，
	size_t corr_bytes = (r_N * 4 - 3) * sizeof(float);
	float *rdp_x_corr = (float*)malloc(corr_bytes);
	float *rdp_y_corr = (float*)malloc(corr_bytes);
	memset(rdp_x_corr, 0, corr_bytes);
	memset(rdp_y_corr, 0, corr_bytes);


	for (int j = 0; j < 3; j++)
	{
		//for (int i = 0; i < n_stream; i++)
		//{
		//	int s_xy_offset = i*n_frame;
		CHECK(hipMemcpy(d_xc, xc, sizeof(float)*n_frame*n_stream, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_yc, yc, sizeof(float)*n_frame*n_stream, hipMemcpyHostToDevice));
		//}
		// 这里测试了一下，把传输XY和计算在一个循环里执行速度不变，但是启动会略微延迟，最终加长总的执行时间

		// 尝试一下分解广度运行来覆盖GPU和CPU的执行时间
		// 第一步，第一次GPU计算
		for (int i = 0; i < n_stream; i++)
		{
			//int s_img_offset = i*n_frame*LEN;
			int s_xy_offset = i*n_frame;
			int s_img_offset = i*n_frame*arraySize;
			int s_rdp_offset = i*n_frame*r_N*t_size;
			compute_radial_profile_tex << <grid2, block2, 0, streams[i] >> >
				(d_xc + s_xy_offset, d_yc + s_xy_offset, d_x_sample, d_y_sample, d_power, t_size, r_N, d_radial_profile + s_rdp_offset, arraySize, s_img_offset);


		}

		for (int i = 0; i < n_stream; i++)
		{
			CHECK(hipStreamSynchronize(streams[i]));
		}

		CHECK(hipMemcpyAsync(radial_profile, d_radial_profile, rdp_bytes*n_frame*n_stream, hipMemcpyDeviceToHost, streams[0])); // 直接全拉回来


																																  //FILE *fr = fopen("r4.txt", "w");
																																  //if (fr == NULL)
																																  //{
																																  //	exit(1);
																																  //}

																																  //for (int t = 0; t < r_N*n_frame; t++)
																																  //{
																																  //	for (int t2 = 0; t2 < t_size; t2++)
																																  //	{
																																  //	fprintf(fr, "%f\t ",check[t2+t*t_size]);
																																  //	}
																																  //	fprintf(fr, "%\n");
																																  //}
																																  //fclose(fr);




		for (int s = 0; s < n_stream; s++)
		{

			int s_rdp_offset = s*n_frame*r_N*t_size;
			int s_xy_offset = s*n_frame;


			int s_rN_offset = s*n_frame*r_N;

			for (int i = 0; i < n_frame; i++)
			{
				int xy_offset = i;
				int rdp_offset = i*r_N*t_size;
				int rN_offset = i*r_N;
				//radial profile 在X Y方向上合并
				getrdp_x_y(radial_profile + rdp_offset + s_rdp_offset, r_N, thetaPerQ, rdp_profile + rN_offset + s_rN_offset, rdp_x, rdp_y, power);


				//get_rev_rdp(rdp_x, rdp_x_rev, 2 * r_N - 1);
				//get_rev_rdp(rdp_y, rdp_y_rev, 2 * r_N - 1);

				//rdp_corr(rdp_x, rdp_x_rev, rdp_x_corr, r_N * 2 - 1);
				//rdp_corr(rdp_y, rdp_y_rev, rdp_y_corr, r_N * 2 - 1);


				self_conv(rdp_x, rdp_x_corr, 2 * r_N - 2);
				self_conv(rdp_y, rdp_y_corr, 2 * r_N - 1);


				// 寻找最大值附近的值及其索引。
				int pkx, pky;
				pkx = findpeak(rdp_x_corr, 2*r_N-2); //4N-3的正中间是2N-1,0-4N-4的正中间应是2N-2
				pky = findpeak(rdp_y_corr, 2*r_N-2);


				// 提取最大值附近的5个点，并计算其下标
				float pkx_value[5] = { 0 };
				float pkx_index[5] = { 0 };
				float pky_value[5] = { 0 };
				float pky_index[5] = { 0 };
				for (int k = 0; k < 5; k++)
				{
					pkx_value[k] = rdp_x_corr[pkx - 2 + k];
					pky_value[k] = rdp_y_corr[pky - 2 + k];
					pkx_index[k] = (-(2 * r_N - 2) + (pkx - 2 + k))*r_step;
					pky_index[k] = (-(2 * r_N - 2) + (pky - 2 + k))*r_step;
				}

				// 接下来分别输入x y方向的相关曲线极值点，并利用最小二乘法计算出修正坐标

				float detx = LeastSquareGuassian(pkx_index, pkx_value, 5);
				float dety = LeastSquareGuassian(pky_index, pky_value, 5);

				//printf("the deviation of center is %f, %f\n", detx, dety);

				xc[i + s_xy_offset] = xc[i + s_xy_offset] + 2 * detx / PI;
				yc[i + s_xy_offset] = yc[i + s_xy_offset] + 2 * dety / PI;

			}
		}
	}

	// 至此已经完成了一轮从图像矩阵到中心坐标的全部过程，剩下的就是迭代QI。
	//printf("good until now!\n");

	for (int i = 0; i < n_stream; i++)
	{
		CHECK(hipStreamDestroy(streams[i]));
	}

	//for (int i = 0; i < n_stream; i++)
	//{
	//	int xy_offset = i*n_frame;
	//	for (int j = 0; j < n_frame; j++)
	//	{
	//		printf("the real center of image is %f, %f\n", xc[j+xy_offset], yc[j+xy_offset]);
	//	}

	//}


	/////// 释放内存空间
	for (int i = 0; i < n_stream; i++)
	{
		int s_mean_offset = i*r_N;
		int s_rN_offset = i*n_frame*r_N;
		for (int j = 0; j < n_frame; j++)
		{
			int rN_offset = j*r_N;
			for (int k = 0; k < r_N; k++)
			{
				p_rdp_mean[k + s_mean_offset] = p_rdp_mean[k + s_mean_offset] + rdp_profile[k + rN_offset + s_rN_offset] / n_frame;
			}
		}

	}
	// 解除纹理绑定！！
	CHECK(hipUnbindTexture(tex));

	//free(check);
	//check = NULL;

	// 释放申请的内存

	free(x_sample);
	x_sample = NULL;
	free(y_sample);
	y_sample = NULL;


	free(radial_profile);
	radial_profile = NULL;

	free(power);
	power = NULL;
	//free(rdp_x_rev);
	//rdp_x_rev = NULL;
	//free(rdp_y_rev);
	//rdp_y_rev = NULL;
	free(rdp_x_corr);
	rdp_x_corr = NULL;
	free(rdp_y_corr);
	rdp_y_corr = NULL;
	free(rdp_x);
	rdp_x = NULL;
	free(rdp_y);
	rdp_y = NULL;


	////
	////
	//CHECK(hipHostFree(d_check));
	CHECK(hipHostFree(d_radial_profile));
	CHECK(hipFree(d_x_sample));
	CHECK(hipFree(d_y_sample));
	CHECK(hipFree(d_power));

	CHECK(hipFree(d_img_mat));
	CHECK(hipFree(d_xc));
	CHECK(hipFree(d_yc));



	return true;
}
