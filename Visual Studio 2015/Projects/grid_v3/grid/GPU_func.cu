#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <conio.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>  // ������������





#define PI 3.141592653


#define BLK_DIM2 32
#define CHECK(call)  \
     { const hipError_t error = call;         \
       if (error != hipSuccess) {            \
            printf("Error:%s:%d, ",__FILE__,__LINE__);  \
            printf("code:%d, reason:%s\n",error,hipGetErrorString(error)); \
            exit(1);     \
          }}   









/*���к������ڼ���ͼ������ĸ����޵�radial profile*/
__global__ void compute_radial_profile_MN
(float *img_array, float *xc, float *yc, float *r_sample, float *t_sample, int t_size, int r_size, float *radial_profile, int arraySize)
{
	const unsigned int idx = (blockIdx.x*blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y*blockDim.y) + threadIdx.y;
	int thread_idx = ((gridDim.x*blockDim.x)*idy) + idx;

	//if (thread_idx > (t_size*r_size*n_frame*n_stream))
	//	return;
	int s = (thread_idx + 1) / (r_size*t_size);  //��i ���ж��ǵڼ���ͼ�����ݣ�Ȼ��ѡȡ��ͬ���������� ����Ӧ�ü�1
	int img_offset = s*arraySize*arraySize;
	int inner_thread_idx = thread_idx % (r_size*t_size);

	int id_r = inner_thread_idx / t_size;
	int id_t = inner_thread_idx % t_size;
	float x = xc[s] + r_sample[id_r] * cos(t_sample[id_t]);
	float y = yc[s] + r_sample[id_r] * sin(t_sample[id_t]);
	int x1 = (int)x;
	int y1 = (int)y;
	int x2 = x1 + 1;
	int y2 = y1 + 1;
	float rdp =
		img_array[y1*arraySize + x1 + img_offset] * ((float)x2 - x)*((float)y2 - y) + \
		img_array[y2*arraySize + x1 + img_offset] * ((float)x2 - x)*(y - (float)y1) + \
		img_array[y1*arraySize + x2 + img_offset] * (x - (float)x1)*((float)y2 - y) + \
		img_array[y2*arraySize + x2 + img_offset] * (x - (float)x1)*(y - (float)y1);
	float power_id = (float)(id_r) / r_size;

	float fall = 1.0 - exp(-(1.0 - power_id)*(1.0 - power_id) / 0.05);
	float rise = 1.0 - exp(-power_id*power_id / 0.01);
	float power = sqrt(fall*rise*power_id * 2.0);
	radial_profile[thread_idx] = rdp*power;
	//d_check[thread_idx] = thread_idx;
}

// ������������������ֵ�õ�Ȩ��ֵ
__global__ void compute_mean_rdp
(float *mean_rdp, float *d_radial_profile)
{
	const unsigned int idx = (blockIdx.x*blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y*blockDim.y) + threadIdx.y;
	int thread_idx = ((gridDim.x*blockDim.x)*idy) + 16 * idx;
	float x_r = 0;
	float y_r = 0;
	float mean = 0;
	for (int i = 0; i < 16; i++)
	{
		mean = mean + d_radial_profile[thread_idx + i];
	}
	mean_rdp[idx] = mean / 16;
}

__global__ void compute_x_rdp
(float *d_radial_profile, float *d_x_rdp, int r_N, int thetaPerQ)
{
	const unsigned int idx = (blockIdx.x*blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y*blockDim.y) + threadIdx.y;
	int thread_idx = ((gridDim.x*blockDim.x)*idy) + thetaPerQ * 4 * idx;
	int s = idx / (2 * r_N - 1);
	int inner_idx = idx%r_N; // ��Ϊ
	float r1 = 0;
	float r2 = 0;
	float r3 = 0;
	float r4 = 0;
	for (int i = 0; i < thetaPerQ; i++)
	{
		r1 = r1 + d_radial_profile[thread_idx + i + s*r_N*thetaPerQ * 4];
		r2 = r2 + d_radial_profile[thread_idx + i + thetaPerQ + s*r_N*thetaPerQ * 4];
		r3 = r3 + d_radial_profile[thread_idx + i + 2 * thetaPerQ + s*r_N*thetaPerQ * 4];
		r4 = r4 + d_radial_profile[thread_idx + i + 3 * thetaPerQ + s*r_N*thetaPerQ * 4];
	}
	// ע�⣬2N+1������λ����N, ��C�����Ǵ�0��ʼ�ģ���������λ��Ӧ����N-1
	d_x_rdp[inner_idx + r_N - 1 + s*(2 * r_N - 1)] = (r1 + r4) / (2 * thetaPerQ);
	d_x_rdp[r_N - 1 - inner_idx + s*(2 * r_N - 1)] = (r2 + r3) / (2 * thetaPerQ);
	//d_y_rdp[inner_idx + r_N-1 + s*(2 * r_N - 1)] = r1+r2;
	//d_y_rdp[r_N-1 - inner_idx + s*(2 * r_N - 1)] = r3+r4;
	//d_mean_rdp[inner_idx +s*r_N] = r1;

}

__global__ void compute_y_rdp
(float *d_radial_profile, float *d_y_rdp, int r_N, int thetaPerQ)
{
	const unsigned int idx = (blockIdx.x*blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y*blockDim.y) + threadIdx.y;
	int thread_idx = ((gridDim.x*blockDim.x)*idy) + thetaPerQ * 4 * idx;
	int s = idx / (2 * r_N - 1);
	int inner_idx = idx%r_N; // ��Ϊidxһ���߳�����������ֵ������r_N���߳̾��Ѿ������2*r_N��ֵ
	float r1 = 0;
	float r2 = 0;
	float r3 = 0;
	float r4 = 0;
	for (int i = 0; i < thetaPerQ; i++)
	{
		r1 = r1 + d_radial_profile[thread_idx + i + s*r_N*thetaPerQ * 4];
		r2 = r2 + d_radial_profile[thread_idx + i + thetaPerQ + s*r_N*thetaPerQ * 4];
		r3 = r3 + d_radial_profile[thread_idx + i + 2 * thetaPerQ + s*r_N*thetaPerQ * 4];
		r4 = r4 + d_radial_profile[thread_idx + i + 3 * thetaPerQ + s*r_N*thetaPerQ * 4];
	}
	// ע�⣬2N+1������λ����N, ��C�����Ǵ�0��ʼ�ģ���������λ��Ӧ����N-1
	//d_x_rdp[inner_idx + r_N-1 + s*(2*r_N-1)] = r1 + r4;
	//d_x_rdp[r_N-1 - inner_idx + s*(2 * r_N - 1)] = r2 + r3;
	d_y_rdp[inner_idx + r_N - 1 + s*(2 * r_N - 1)] = (r1 + r2) / (2 * thetaPerQ);
	d_y_rdp[r_N - 1 - inner_idx + s*(2 * r_N - 1)] = (r3 + r4) / (2 * thetaPerQ);
	//d_mean_rdp[inner_idx +s*r_N] = r1;

}
// �Ծ����������һ��A
__global__ void conv_Kernelx(const float *A, float *d_conv_xy, const int rx_N, const int n_frame) {
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	int s = idx / (2 * rx_N - 1);
	int inner_idx = idx % (2 * rx_N - 1); // һ���߳�һ���㣬��Ҫ��ô����㡣
	if (idx < n_frame*(2 * rx_N) - 1) {
		float my_sum = 0;
		for (int i = 0; i < rx_N; i++)
			if (((inner_idx < rx_N) && (i <= inner_idx)) || ((inner_idx >= rx_N) && (i >(inner_idx - rx_N)))) my_sum += A[i + s*rx_N] * A[inner_idx - i + s*rx_N];
		d_conv_xy[idx] = my_sum;
	}
}
// �Ծ����������һ��A
__global__ void conv_Kernely(const float *A, float *d_conv_xy, const int rx_N, const int n_frame) {
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	int s = idx / (2 * rx_N - 1);
	int inner_idx = idx % (2 * rx_N - 1); // һ���߳�һ���㣬��Ҫ��ô����㡣
	if (idx < n_frame*(2 * rx_N) - 1) {
		float my_sum = 0;
		for (int i = 0; i < rx_N; i++)
			if (((inner_idx < rx_N) && (i <= inner_idx)) || ((inner_idx >= rx_N) && (i >(inner_idx - rx_N)))) my_sum += A[i + s*rx_N] * A[inner_idx - i + s*rx_N];
		d_conv_xy[idx] = my_sum;
	}
}

__global__ void compute_radial_profile
(float *img_array, float xc, float yc, float *r_sample, float *t_sample, int t_size, int r_size, float *radial_profile, int arraySize)
{
	const unsigned int idx = (blockIdx.x*blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y*blockDim.y) + threadIdx.y;
	int thread_idx = ((gridDim.x*blockDim.x)*idy) + idx;
	//int i = thread_idx / (arraySize*arraySize); 

	//int inner_thread_idx = thread_idx % (arraySize*arraySize);
	int id_r = thread_idx / t_size;
	int id_t = thread_idx % t_size;
	float x = xc + r_sample[id_r] * sin(t_sample[id_t]);
	float y = yc + r_sample[id_r] * cos(t_sample[id_t]);
	int x1 = (int)x;
	int y1 = (int)y;
	int x2 = x1 + 1;
	int y2 = y1 + 1;
	float rdp = img_array[y1*arraySize + x1] * ((float)x2 - x)*((float)y2 - y) + \
		img_array[y2*arraySize + x1] * ((float)x2 - x)*(y - (float)y1) + \
		img_array[y1*arraySize + x2] * (x - (float)x1)*((float)y2 - y) + \
		img_array[y2*arraySize + x2] * (x - (float)x1)*(y - (float)y1);
	float power_id = id_r / (float)r_size;

	float fall = 1.0 - exp(-(1.0 - power_id)*(1.0 - power_id) / 0.05);
	float rise = 1.0 - exp(-power_id*power_id / 0.01);
	float power = sqrt(fall*rise*power_id * 2.0);
	radial_profile[thread_idx] = rdp*power;

}