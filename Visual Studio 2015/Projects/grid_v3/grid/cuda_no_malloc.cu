#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <conio.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>  // ������������





#define PI 3.141592653

#define BLK_DIM 256

#define CHECK(call)  \
     { const hipError_t error = call;         \
       if (error != hipSuccess) {            \
            printf("Error:%s:%d, ",__FILE__,__LINE__);  \
            printf("code:%d, reason:%s\n",error,hipGetErrorString(error)); \
            exit(1);     \
          }}   

//void * alloc(size_t size)
//{
//	void *new_mem;
//	new_mem = malloc(size);
//	if (new_mem == NULL)
//	{
//		printf("Out of memory!\n");
//		exit(1);
//
//	}
//	return new_mem;
//}

/*Declare statically six arrays of ARRAY_SIZE each
** ����CPU����������Լ�������ŷ��ؽ���ľ���
//*/
//float cpu_cen_mat_x[ARRAY_SIZE_Y][ARRAY_SIZE_X];
//float cpu_cen_mat_y[ARRAY_SIZE_Y][ARRAY_SIZE_X];
//float cpu_yc[1];
//float cpu_xc[1];
//float img_mat[ARRAY_SIZE_Y][ARRAY_SIZE_X] = { 0 };  // ������ó�ʼ��һ�£� ��Ȼ���ױ���


int compInc(const void *a, const void *b);







/*���к������ڼ���ͼ������ĸ����޵�radial profile*/
__global__ void compute_radial_profile2
(float *img_array, float *xc, float *yc, float *r_sample, float *t_sample, int t_size, int r_size, float *radial_profile, int arraySize);






// �´���һ�������ֵ�ĺ���
void getMean(float *img_mat, float *arrayMed, int len, int n_stream);



void getrdp_x_y(float *rdp_matrix, int r_N, int theta_num_perQ, float *rdp_all, float *rdp_x, float *rdp_y);

/*���º��������õ�radial profile �ķ�ת��Ϊ�������׼��*/
void get_rev_rdp(float *rdp, float *rdp_rev, int rdp_x_size);

/*���º������������������еĻ�������ߣ�����rdp_corr�����ս����N��rdp�Ĵ�С*/
void rdp_corr(float *rdp, float *rdp_rev, float *rdp_corr, int N);


/*���º�������Ѱ�����ߵķ�ֵ������������λ��*/
float findpeak(float *rdp_corr, int N);



float LeastSquareGuassian(float *arr_x, float *arr_y, int arr_N);



void getCentroid(float *p_img, int arraySize, int n_stream, float *xc, float *yc);


void getCentroid3(float *p_img, int arraySize, int n_stream, float *xc, float *yc)
{
	int LEN = arraySize*arraySize;
	float temp = 0;
	float A = 0;
	float *array_med3 = (float*)malloc(n_stream*sizeof(float));
	getMean(p_img, array_med3, LEN, n_stream);
	for (int i = 0; i < n_stream; i++)
	{
		int i_offset = i*LEN;
		temp = 0;
		A = 0;
		xc[i] = 0;
		yc[i] = 0;
		for (int j = i_offset; j < LEN+i_offset; j++)
		{
			int arr_x = (j-i_offset) % arraySize;
			int arr_y = (j-i_offset) / arraySize;
			float temp = p_img[j] - array_med3[i];
			temp = fabs(temp);
			A += temp;
			xc[i] += (temp)*arr_x;
			yc[i] += (temp)*arr_y;
		}
		xc[i] = xc[i] / A;
		yc[i] = yc[i] / A;
		printf("the center of array is %f, %f\n", xc[i], yc[i]);

	}


	free(array_med3);
	//*array_med3 = NULL;


}









/////////////////////////////////////////////////////////////////////////////////////////////////////
void cuda_QI3(float *p_img, int arraySize, int n_stream, float *rdp_profile, float *xc, float *yc)
{
	//int arraySize = 80;

	size_t img_bytes = arraySize*arraySize * sizeof(float);   // ����Ԫ����ռ�ռ�
	size_t f_bytes = sizeof(float);

	//hipStream_t *streams = (hipStream_t *)malloc(n_stream * sizeof(hipStream_t));

	int LEN = arraySize*arraySize;                            // ͼ�����Ԫ����
	int full_LEN = LEN*n_stream;

	//const dim3 block(BLK_DIM);
	//const dim3 grid((full_LEN + block.x - 1) / block.x, 1);
	//float *p_img = &img_mat[0][0][0];

	// ����getCentroid ��������n_stream ��ͼ������xc,yc

	getCentroid3(p_img, arraySize, n_stream, xc, yc);
	printf("please be good!\n");


	/// ������ʵ��QI�㷨  
	// �ȶ�����صı�����
	float r_step = 0.4;
	int theta_num_perQ = 8;
	int t_size = theta_num_perQ * 4;
	int r_max = arraySize / 2 - 2;
	int r_N = r_max / r_step;

	// ���������㣬��Щ��������е�ROI������һ����
	size_t r_bytes = r_N * sizeof(float);
	size_t t_bytes = t_size * sizeof(float);
	float *r_sample = (float *)malloc(r_bytes);
	if (*r_sample = NULL)
		printf("out of memory!");

	float *t_sample = (float *)malloc(t_bytes);
	if (*t_sample = NULL)
		printf("out of memory!");
	printf("good now!");
	for (int i = 0; i < r_N; i++)
	{
		r_sample[i] = i*r_step;
		//printf("the r_sequence is :%f\n", r_sample[i]);
	}
	for (int i = 0; i < theta_num_perQ * 4; i++)
	{
		t_sample[i] = i*PI * 2 / (theta_num_perQ * 4);
		//printf("the t_sequence is :%f\n", t_sample[i]);
	}

	//Ԥ����QI�˺�����ؽ���������Լ���Ӧ��GPU����
	size_t rdp_bytes = r_N*theta_num_perQ * 4 * sizeof(float);
	size_t rdp_s_bytes = rdp_bytes*n_stream;
	//size_t rN_bytes = r_N * sizeof(float);


	float * radial_profile = (float *)malloc(rdp_s_bytes);  // �������radial profile������

	// GPU allocation
	float *d_radial_profile = NULL;
	float *d_r_sample = NULL;
	float *d_t_sample = NULL;
	float *d_img_mat = NULL;
	float *d_xc = NULL;
	float *d_yc = NULL;
	////

	CHECK(hipMalloc(&d_radial_profile, rdp_s_bytes));
	CHECK(hipMalloc(&d_r_sample, r_bytes));
	CHECK(hipMalloc(&d_t_sample, t_bytes));
	CHECK(hipMalloc(&d_img_mat, img_bytes*n_stream));
	CHECK(hipMalloc(&d_xc, sizeof(float)*n_stream));
	CHECK(hipMalloc(&d_yc, sizeof(float)*n_stream));



	// ��������  
	//CHECK(hipMemcpy(d_radial_profile, radial_profile, rdp_bytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_r_sample, r_sample, r_bytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_t_sample, t_sample, t_bytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_img_mat, p_img, img_bytes*n_stream, hipMemcpyHostToDevice));

	//for (int i = 0; i < n_stream; i++)
	//{
	//	CHECK(hipStreamSynchronize(streams[i]));
	//}
	int rdp_LEN = r_N*t_size*n_stream;
	const dim3 block2(BLK_DIM);
	const dim3 grid2((rdp_LEN + block2.x - 1) / block2.x, 1);

	//printf("grid.x %d, grid.y %d,grid.z,%d\n",grid2.x,grid2.y,grid2.z);
	//printf("block.x %d, block.y %d,block.z,%d\n", block2.x, block2.y, block2.z);

	size_t rdpxy_byte = (r_N*theta_num_perQ * 4 * 2 - 1) * sizeof(float);
	float *rdp_x = (float *)malloc(rdpxy_byte);
	float *rdp_y = (float *)malloc(rdpxy_byte);

	// ������ȡrdp_x �ķ����������η�������
	float *rdp_x_rev = (float *)malloc(rdpxy_byte);
	float *rdp_y_rev = (float *)malloc(rdpxy_byte);

	// ���������л���ؼ��㣬
	size_t corr_bytes = (r_N*theta_num_perQ * 4 * 4 - 3) * sizeof(float);
	float *rdp_x_corr = (float*)malloc(corr_bytes);
	float *rdp_y_corr = (float*)malloc(corr_bytes);



	for (int j = 0; j < 3; j++)
	{
		CHECK(hipMemcpy(d_xc, xc, sizeof(float)*n_stream, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_yc, yc, sizeof(float)*n_stream, hipMemcpyHostToDevice));

		compute_radial_profile2 << <grid2, block2 >> >
			(d_img_mat, d_yc, d_xc, d_r_sample, d_t_sample, t_size, r_N, d_radial_profile, arraySize);


		CHECK(hipMemcpy(radial_profile, d_radial_profile, rdp_s_bytes, hipMemcpyDeviceToHost));


		for (int i = 0; i < n_stream; i++)
		{
			int rdp_offset = i*r_N*theta_num_perQ * 4;
			int rN_offset = i*r_N;
			// radial profile ��X Y�����Ϻϲ�
			getrdp_x_y( radial_profile + rdp_offset, r_N, theta_num_perQ, rdp_profile+rN_offset, rdp_x, rdp_y);

			get_rev_rdp(rdp_x, rdp_x_rev, 2 * r_N - 1);

			get_rev_rdp(rdp_y, rdp_y_rev, 2 * r_N - 1);

			rdp_corr(rdp_x, rdp_x_rev, rdp_x_corr, r_N * 2 - 1);
			rdp_corr(rdp_y, rdp_y_rev, rdp_y_corr, r_N * 2 - 1);

			// Ѱ�����ֵ������ֵ����������
			int pkx, pky;
			pkx = findpeak(rdp_x_corr, r_N * 4 - 3);
			pky = findpeak(rdp_y_corr, r_N * 4 - 3);

			// ��ȡ���ֵ������5���㣬���������±�
			float pkx_value[5] = { 0 };
			float pkx_index[5] = { 0 };
			float pky_value[5] = { 0 };
			float pky_index[5] = { 0 };
			for (int k = 0; k < 5; k++)
			{
				pkx_value[k] = rdp_x_corr[pkx - 2 + k];
				pky_value[k] = rdp_y_corr[pky - 2 + k];
				pkx_index[k] = (-(2 * r_N - 2) + (pkx - 2 + k))*r_step;
				pky_index[k] = (-(2 * r_N - 2) + (pky - 2 + k))*r_step;
			}

			// �������ֱ�����x y�����������߼�ֵ�㣬��������С���˷��������������

			float detx = LeastSquareGuassian(pkx_index, pkx_value, 5);
			float dety = LeastSquareGuassian(pky_index, pky_value, 5);

			printf("the deviation of center is %f, %f\n", detx, dety);

			xc[i] = xc[i] - 2 * detx / PI;
			yc[i] = yc[i] - 2 * dety / PI;
		}
	}
	// �����Ѿ������һ�ִ�ͼ��������������ȫ�����̣�ʣ�µľ��ǵ���QI��
	//printf("good until now!\n");



	for (int i = 0; i < n_stream; i++)
	{
		printf("the real center of image is %f, %f\n", xc[i], yc[i]);
	}


	/////// �ͷ��ڴ�ռ�

	free(r_sample);
	//*r_sample = NULL;
	free(t_sample);
	//*t_sample = NULL;

	free(radial_profile);
	//*radial_profile = NULL;
	free(rdp_x);
	//*rdp_x = NULL;
	free(rdp_y);
	//*rdp_y = NULL;

	free(rdp_x_rev);
	//*rdp_x_rev = NULL;
	free(rdp_y_rev);
	//*rdp_y_rev = NULL;
	free(rdp_x_corr);
	//*rdp_x_corr = NULL;
	free(rdp_y_corr);
	//*rdp_y_corr = NULL;




	////
	////
	CHECK(hipFree(d_radial_profile));
	CHECK(hipFree(d_r_sample));
	CHECK(hipFree(d_t_sample));
	CHECK(hipFree(d_img_mat));
	CHECK(hipFree(d_xc));
	CHECK(hipFree(d_yc));

}


