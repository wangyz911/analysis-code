#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <conio.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>  


#define PI 3.141592653
#define BLK_DIM 256
#define CHECK(call)  \
     { const hipError_t error = call;         \
       if (error != hipSuccess) {            \
            printf("Error:%s:%d, ",__FILE__,__LINE__);  \
            printf("code:%d, reason:%s\n",error,hipGetErrorString(error)); \
            exit(1);     \
          }}

int compInc(const void *a, const void *b);
//{
//	return *(int *)a - *(int *)b;
//}

__global__ void compute_radial_profile
(float *img_array, float xc, float yc, float *r_sample, float *t_sample, int t_size, int r_size, float *radial_profile, int arraySize);
//{
//	const unsigned int idx = (blockIdx.x*blockDim.x) + threadIdx.x;
//	const unsigned int idy = (blockIdx.y*blockDim.y) + threadIdx.y;
//	int thread_idx = ((gridDim.x*blockDim.x)*idy) + idx;
//	//int i = thread_idx / (arraySize*arraySize); 
//
//	//int inner_thread_idx = thread_idx % (arraySize*arraySize);
//	int id_r = thread_idx / t_size;
//	int id_t = thread_idx % t_size;
//	float x = xc + r_sample[id_r] * sin(t_sample[id_t]);
//	float y = yc + r_sample[id_r] * cos(t_sample[id_t]);
//	int x1 = (int)x;
//	int y1 = (int)y;
//	int x2 = x1 + 1;
//	int y2 = y1 + 1;
//	float rdp = img_array[y1*arraySize + x1] * ((float)x2 - x)*((float)y2 - y) + \
//		img_array[y2*arraySize + x1] * ((float)x2 - x)*(y - (float)y1) + \
//		img_array[y1*arraySize + x2] * (x - (float)x1)*((float)y2 - y) + \
//		img_array[y2*arraySize + x2] * (x - (float)x1)*(y - (float)y1);
//	float power_id = id_r/ (float)r_size;
//
//	float fall = 1.0 - exp(-(1.0 - power_id)*(1.0 - power_id) / 0.05);
//	float rise = 1.0 - exp(-power_id*power_id / 0.01);
//	float power = sqrt(fall*rise*power_id * 2.0);
//	radial_profile[thread_idx] = rdp*power;
//
//}

void getMean(float *img_mat, float *arrayMed, int len, int n_stream);
//{
//	for (int j = 0; j < n_stream; j++)
//	{
//		arrayMed[j] = 0;
//		for (int i = j*len; i < (j + 1)*len; i++)
//		{
//			arrayMed[j] += img_mat[i];
//		}
//		arrayMed[j] = arrayMed[j] / len;
//	}
//
//}



void getrdp_x_y(float *rdp_matrix, int r_N, int theta_num_perQ, float *rdp_all, float *rdp_x, float *rdp_y);
//{
//	size_t r_byte = sizeof(float)*r_N;
//	float *rdp_1 = (float *)malloc(r_byte);
//	float *rdp_2 = (float *)malloc(r_byte);
//	float *rdp_3 = (float *)malloc(r_byte);
//	float *rdp_4 = (float *)malloc(r_byte);
//
//	for (int i = 0; i < r_N; i++)
//	{
//		rdp_1[i] = 0;
//		rdp_2[i] = 0;
//		rdp_3[i] = 0;
//		rdp_4[i] = 0;
//
//		for (int j = 0; j < theta_num_perQ; j++)
//		{
//			rdp_1[i] = rdp_1[i] + rdp_matrix[j + 4 * i*theta_num_perQ];
//
//			rdp_2[i] = rdp_2[i] + rdp_matrix[j + 1 * theta_num_perQ + 4 * i*theta_num_perQ];
//
//			rdp_3[i] = rdp_3[i] + rdp_matrix[j + 2 * theta_num_perQ + 4 * i*theta_num_perQ];
//
//			rdp_4[i] = rdp_4[i] + rdp_matrix[j + 3 * theta_num_perQ + 4 * i*theta_num_perQ];
//		};
//		rdp_all[i] = (rdp_1[i] + rdp_2[i] + rdp_3[i] + rdp_4[i]) / (4*theta_num_perQ);
//	};
//
//	for (int k = 0; k < (2 * r_N - 1); k++)
//	{
//		if (k < r_N - 1)
//		{
//			rdp_x[k] = (rdp_2[r_N - 1 - k] + rdp_3[r_N - 1 - k])/theta_num_perQ;
//			rdp_y[k] = (rdp_3[r_N - 1 - k] + rdp_4[r_N - 1 - k])/theta_num_perQ;
//		}
//		else
//		{
//			rdp_x[k] = (rdp_1[k + 1 - r_N] + rdp_4[k + 1 - r_N]) /theta_num_perQ;
//			rdp_y[k] = (rdp_1[k + 1 - r_N] + rdp_2[k + 1 - r_N]) /theta_num_perQ;
//		}
//	}
//
//	free(rdp_1);
//	rdp_1 = NULL;
//	free(rdp_2);
//	rdp_2 = NULL;
//	free(rdp_3);
//	rdp_3 = NULL;
//	free(rdp_4);
//	rdp_4 = NULL;
//
//
//}

void get_rev_rdp(float *rdp, float *rdp_rev, int rdp_x_size);
//{
//	for (int i = 0; i < rdp_x_size; i++)
//	{
//		rdp_rev[i] = rdp[rdp_x_size - 1 - i];
//	}
//	return;
//}

void rdp_corr(float *rdp, float *rdp_rev, float *rdp_corr, int N);
//{
//	float corr_ij;
//	int    delay, i, j;
//
//	for (delay = -N + 1; delay < N; delay++)
//	{
//		//Calculate the numerator
//		corr_ij = 0;
//		for (i = 0; i < N; i++)
//		{
//			j = i + delay;
//			if ((j < 0) || (j >= N))  //The series are no wrapped,so the value is ignored
//				continue;
//			else
//				corr_ij += (rdp[i] * rdp_rev[j]);
//		}
//
//		//Calculate the correlation series at "delay"
//		rdp_corr[delay + N - 1] = corr_ij;
//	}
//}

float findpeak(float *rdp_corr, int N);
//{
//	float max = rdp_corr[0];
//	int index = 0;
//	for (int i = 0; i < N; i++)
//	{
//		if (max <= rdp_corr[i])
//		{
//			index = i;
//			max = rdp_corr[i];
//		}
//		;
//	}
//	return index;
//}

float LeastSquareGuassian(float *arr_x, float *arr_y, int arr_N);
//{
//	const int rank_ = 2;
//	float atemp[2 * (rank_ + 1)] = { 0 }, b[rank_ + 1] = { 0 }, a[rank_ + 1][rank_ + 1];
//	int i, j, k;
//
//	for (i = 0; i < arr_N; i++) {  //
//		atemp[1] += arr_x[i];
//		atemp[2] += pow(arr_x[i], 2);
//		atemp[3] += pow(arr_x[i], 3);
//		atemp[4] += pow(arr_x[i], 4);
//		//atemp[5] += pow(arr_x[i], 5);
//		//atemp[6] += pow(arr_x[i], 6);
//		b[0] += arr_y[i];
//		b[1] += arr_x[i] * arr_y[i];
//		b[2] += pow(arr_x[i], 2) * arr_y[i];
//		//b[3] += pow(arr_x[i], 3) * arr_y[i];
//	}
//
//	atemp[0] = arr_N;
//	/*
//	for(i = 0; i <= 2 * rank_; i++)  printf("atemp[%d] = %f\n", i, atemp[i]);
//	printf("\n");
//	for(i = 0; i <= rank_; i++)  printf("b[%d] = %f\n", i, b[i]);
//	printf("\n");
//	*/
//	for (i = 0; i < rank_ + 1; i++) {
//		k = i;
//		for (j = 0; j < rank_ + 1; j++)  a[i][j] = atemp[k++];
//	}
//	/*
//	for(i = 0; i < rank_ + 1; i++){
//	for(j = 0; j < rank_ + 1; j++)  printf("a[%d][%d] = %-17f  ", i, j, a[i][j]);
//	printf("\n");
//	}
//	printf("\n");
//	*/
//
//
//	for (k = 0; k < rank_ + 1 - 1; k++) {
//		int column = k;
//		float mainelement = a[k][k];
//
//		for (i = k; i < rank_ + 1; i++)
//			if (fabs(a[i][k]) > mainelement) {
//				mainelement = fabs(a[i][k]);
//				column = i;
//			}
//		for (j = k; j < rank_ + 1; j++)
//		{
//			float atemp = a[k][j];
//			a[k][j] = a[column][j];
//			a[column][j] = atemp;
//		}
//		float btemp = b[k];
//		b[k] = b[column];
//		b[column] = btemp;
//
//		for (i = k + 1; i < rank_ + 1; i++) {
//			float Mik = a[i][k] / a[k][k];
//			for (j = k; j < rank_ + 1; j++)  a[i][j] -= Mik * a[k][j];
//			b[i] -= Mik * b[k];
//		}
//	}
//	/*
//	for(i = 0; i < rank_ + 1; i++){
//	for(j = 0; j < rank_ + 1; j++)  printf("%20f", a[i][j]);
//	printf("%20f\n", b[i]);
//	}
//	printf("\n");
//	*/
//	b[rank_ + 1 - 1] /= a[rank_ + 1 - 1][rank_ + 1 - 1];
//	for (i = rank_ + 1 - 2; i >= 0; i--) {
//		float sum = 0;
//		for (j = i + 1; j < rank_ + 1; j++)  sum += a[i][j] * b[j];
//		b[i] = (b[i] - sum) / a[i][i];
//	}
//
//
//	/*printf("P(x) = %f + %f x + %f x^2\n", b[0], b[1], b[2]);*/
//	float detx = -b[1] / (2 * b[2]);
//	return detx;
//
//
//}

void getCentroid(float *p_img, int arraySize, int n_stream, float *xc, float *yc);
//{
//	int LEN = arraySize*arraySize;
//	float temp = 0;
//	float A = 0;
//	float *array_med3 = (float*)malloc(n_stream * sizeof(float));
//	getMean(p_img, array_med3, LEN, n_stream);
//	for (int i = 0; i < n_stream; i++)
//	{
//		int i_offset = i*LEN;
//		temp = 0;
//		A = 0;
//		xc[i] = 0;
//		yc[i] = 0;
//		for (int j = i_offset; j < LEN + i_offset; j++)
//		{
//			int arr_x = (j - i_offset) % arraySize;
//			int arr_y = (j - i_offset) / arraySize;
//			float temp = p_img[j] - array_med3[i];
//			temp = fabs(temp);
//			A += temp;
//			xc[i] += (temp)*arr_x;
//			yc[i] += (temp)*arr_y;
//		}
//		xc[i] = xc[i] / A;
//		yc[i] = yc[i] / A;
//		//printf("the center of array is %f, %f\n", xc[i], yc[i]);
//
//	}
//
//
//	free(array_med3);
//	array_med3 = NULL;
//}





void cuda_QI(float *p_img, int arraySize, int n_stream, float *rdp_profile, float *xc, float *yc)
{
	size_t img_bytes = arraySize*arraySize * sizeof(float);
	size_t f_bytes = sizeof(float);

	hipStream_t *streams = (hipStream_t *)malloc(n_stream * sizeof(hipStream_t));

	int LEN = arraySize*arraySize;

	//float *p_img = &img_mat[0][0][0];
	getCentroid(p_img, arraySize, n_stream, xc, yc);

	float r_step = 0.4;
	int theta_num_perQ = 8;
	int t_size = theta_num_perQ * 4;
	int r_max = arraySize / 2 - 2;
	int r_N = r_max / r_step;

	int rdp_LEN = r_N*t_size;
	const dim3 block2(BLK_DIM);
	const dim3 grid2((rdp_LEN + block2.x - 1) / block2.x, 1);

	size_t r_bytes = r_N * sizeof(float);
	size_t t_bytes = theta_num_perQ * 4 * sizeof(float);
	float *r_sample = (float *)malloc(r_bytes);
	float *t_sample = (float *)malloc(t_bytes);
	for (int i = 0; i < r_N; i++)
	{
		r_sample[i] = i*r_step;
	}
	for (int i = 0; i < theta_num_perQ * 4; i++)
	{
		t_sample[i] = i*PI * 2 / (theta_num_perQ * 4);
	}

	size_t rdp_bytes = r_N*theta_num_perQ * 4 * sizeof(float);
	float * radial_profile = (float *)malloc(rdp_bytes*n_stream);
	float *d_radial_profile = NULL;
	float *d_r_sample = NULL;
	float *d_t_sample = NULL;
	float *d_img_mat = NULL;

	CHECK(hipHostMalloc(&d_radial_profile, rdp_bytes*n_stream));
	CHECK(hipHostMalloc(&d_r_sample, r_bytes));
	CHECK(hipHostMalloc(&d_t_sample, t_bytes));
	CHECK(hipHostMalloc(&d_img_mat, img_bytes*n_stream));

	for (int i = 0; i < n_stream; i++)
	{
		CHECK(hipStreamCreate(&streams[i]));
	}

	for (int i = 0; i < n_stream; i++)
	{
		int offset = i*LEN;
		CHECK(hipMemcpyAsync(d_r_sample, r_sample, r_bytes, hipMemcpyHostToDevice, streams[i]));
		CHECK(hipMemcpyAsync(d_t_sample, t_sample, t_bytes, hipMemcpyHostToDevice, streams[i]));
		CHECK(hipMemcpyAsync(d_img_mat + offset, p_img + offset, img_bytes, hipMemcpyHostToDevice, streams[i]));
	}

	size_t rdpxy_byte = (r_N*theta_num_perQ * 4 * 2 - 1) * sizeof(float);
	float *rdp_x = (float *)malloc(rdpxy_byte);
	float *rdp_y = (float *)malloc(rdpxy_byte);

	float *rdp_x_rev = (float *)malloc(rdpxy_byte);
	float *rdp_y_rev = (float *)malloc(rdpxy_byte);

	size_t corr_bytes = (r_N*theta_num_perQ * 4 * 4 - 3) * sizeof(float);
	float *rdp_x_corr = (float*)malloc(corr_bytes);
	float *rdp_y_corr = (float*)malloc(corr_bytes);

	for (int j = 0; j < 3; j++)
	{
		for (int i = 0; i < n_stream; i++)
		{
			int offset = i*LEN;
			int rdp_offset = i*r_N*theta_num_perQ * 4;
			int xy_offset = i;

			compute_radial_profile << <grid2.x, block2, 0, streams[i] >> >
				(d_img_mat + offset, *(yc + xy_offset), *(xc + xy_offset), d_r_sample, d_t_sample, t_size, r_N, d_radial_profile + rdp_offset, arraySize);
		}
		for (int i = 0; i < n_stream; i++)
		{
			CHECK(hipStreamSynchronize(streams[i]));
		}

		for (int i = 0; i < n_stream; i++)
		{
			int rdp_offset = i*r_N*theta_num_perQ * 4;
			CHECK(hipMemcpyAsync(radial_profile + rdp_offset, d_radial_profile + rdp_offset, rdp_bytes, hipMemcpyDeviceToHost, streams[i]));
		}

		for (int i = 0; i < n_stream; i++)
		{
			int rdp_offset = i*r_N*theta_num_perQ * 4;

			getrdp_x_y(radial_profile + rdp_offset, r_N, theta_num_perQ, rdp_profile, rdp_x, rdp_y);

			get_rev_rdp(rdp_x, rdp_x_rev, 2 * r_N - 1);
			get_rev_rdp(rdp_y, rdp_y_rev, 2 * r_N - 1);

			rdp_corr(rdp_x, rdp_x_rev, rdp_x_corr, r_N * 2 - 1);
			rdp_corr(rdp_y, rdp_y_rev, rdp_y_corr, r_N * 2 - 1);

			int pkx, pky;
			pkx = findpeak(rdp_x_corr, r_N * 4 - 3);
			pky = findpeak(rdp_y_corr, r_N * 4 - 3);

			float pkx_value[5] = { 0 };
			float pkx_index[5] = { 0 };
			float pky_value[5] = { 0 };
			float pky_index[5] = { 0 };
			for (int k = 0; k < 5; k++)
			{
				pkx_value[k] = rdp_x_corr[pkx - 2 + k];
				pky_value[k] = rdp_y_corr[pky - 2 + k];
				pkx_index[k] = (-(2 * r_N - 2) + (pkx - 2 + k))*r_step;
				pky_index[k] = (-(2 * r_N - 2) + (pky - 2 + k))*r_step;
			}

			float detx = LeastSquareGuassian(pkx_index, pkx_value, 5);
			float dety = LeastSquareGuassian(pky_index, pky_value, 5);

			xc[i] = xc[i] - 2 * detx / PI;
			yc[i] = yc[i] - 2 * dety / PI;
		}
	}

	for (int i = 0; i < n_stream; i++)
	{
		CHECK(hipStreamDestroy(streams[i]));
	}
	free(r_sample);
	r_sample = NULL;
	free(t_sample);
	t_sample = NULL;
	free(radial_profile);
	radial_profile = NULL;
	free(rdp_x);
	rdp_x = NULL;
	free(rdp_y);
	rdp_y = NULL;
	free(rdp_x_rev);
	rdp_x_rev = NULL;
	free(rdp_y_rev);
	rdp_y_rev = NULL;
	free(rdp_x_corr);
	rdp_x_corr = NULL;
	free(rdp_y_corr);
	rdp_y_corr = NULL;


	CHECK(hipHostFree(d_radial_profile));
	CHECK(hipHostFree(d_r_sample));
	CHECK(hipHostFree(d_t_sample));
	CHECK(hipHostFree(d_img_mat));


}
