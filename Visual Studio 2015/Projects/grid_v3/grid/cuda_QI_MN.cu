#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <conio.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>  // ������������
#include "gpu_func.cuh"





#define PI 3.141592653


#define BLK_DIM2 32
#define CHECK(call)  \
     { const hipError_t error = call;         \
       if (error != hipSuccess) {            \
            printf("Error:%s:%d, ",__FILE__,__LINE__);  \
            printf("code:%d, reason:%s\n",error,hipGetErrorString(error)); \
            exit(1);     \
          }}   



// Texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex;

__global__ void compute_radial_profile_tex
(float *d_xc, float *d_yc, float *d_x_sample, float *d_y_sample, float *power, int t_size, int r_size, float *d_radial_profile, int arraySize)
{
	const unsigned int idx = (blockIdx.x*blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y*blockDim.y) + threadIdx.y;
	int thread_idx = ((gridDim.x*blockDim.x)*idy) + idx;
	int s = thread_idx / (r_size*t_size);  //��i ���ж��ǵڼ���ͼ�����ݣ�Ȼ��ѡȡ��ͬ���������� ���ﲻӦ�ü�1
	int img_offset = s*arraySize*arraySize;
	int inner_thread_idx = thread_idx % (r_size*t_size);
	int p_id = inner_thread_idx/t_size;
	float x = d_xc[s] + d_x_sample[inner_thread_idx];
	float y = d_yc[s] + d_y_sample[inner_thread_idx]+s*arraySize;
	float p = power[p_id];
	d_radial_profile[thread_idx] = tex2D(tex, x + 0.5f, y+ 0.5f)*p;
	//d_check[thread_idx] = d_radial_profile[thread_idx];
}
//// ׼��ÿ���˺������һ�㣬���Ǻ�������׳���
//__global__ void compute_radial_profile_tex_4
//(float *d_xc, float *d_yc, float *d_x_sample, float *d_y_sample, float *power, int t_size, int r_size, float *d_radial_profile, int arraySize, float *d_check, int n_frame)
//{
//
//	const unsigned int idx = (blockIdx.x*blockDim.x) + threadIdx.x;
//	//const unsigned int idy = (blockIdx.y*blockDim.y) + threadIdx.y;
//	if (idx<= r_size*t_size*n_frame)
//	{
//		//int thread_idx = ((gridDim.x*blockDim.x)*idy) + 4*idx;
//		int thread_idx = 4 * idx;
//		int s = thread_idx / (r_size*t_size);  //��i ���ж��ǵڼ���ͼ�����ݣ�Ȼ��ѡȡ��ͬ���������� ���ﲻӦ�ü�1
//		int img_offset = s*arraySize*arraySize;
//		int inner_thread_idx = thread_idx % (r_size*t_size);
//		int p_id = inner_thread_idx / t_size;
//		for (int i =0; i < 4; i++)
//		{
//			float x = d_xc[s] + d_x_sample[inner_thread_idx+i];
//			float y = d_yc[s] + d_y_sample[inner_thread_idx+i] + s*arraySize;
//			float p = power[p_id];
//			d_radial_profile[thread_idx+i] = tex2D(tex, x + 0.5f, y + 0.5f)*p;
//			d_check[thread_idx + i] = d_radial_profile[thread_idx + i];
//		}
//
//	}
//
//}


/////////////////////////////////////////////////////////////////////////////////////////////////////
void cuda_QI4(float *p_img_4d, int arraySize, int n_frame,int n_stream, float *rdp_profile, float *xc, float *yc)
{
	//int arraySize = 80;

	size_t img_bytes = arraySize*arraySize * sizeof(float);   // ����Ԫ����ռ�ռ�
	size_t f_bytes = sizeof(float);

	//hipStream_t *streams = (hipStream_t *)malloc(n_stream * sizeof(hipStream_t));

	int LEN = arraySize*arraySize;                            // ͼ�����Ԫ����
	int s_LEN = LEN*n_frame;

	// ����getCentroid ��������n_stream ��ͼ������xc,yc

	getCentroid_MN(p_img_4d, arraySize, n_frame,n_stream, xc, yc);
	//printf("please be good!\n");


	/// ������ʵ��QI�㷨  
	// �ȶ�����صı�����
	float r_step = 0.4;
	int thetaPerQ =4;
	int t_size = thetaPerQ * 4;
	//int t_size =4 ;
	int r_max = arraySize / 2 - 5;
	int r_N = r_max / r_step;

	// ���������㣬��Щ��������е�ROI������һ����
	size_t r_bytes = r_N * sizeof(float);
	size_t t_bytes = t_size * sizeof(float);



	//Ԥ����QI�˺�����ؽ���������Լ���Ӧ��GPU����
	size_t rdp_bytes = r_N*t_size * sizeof(float);
	size_t rdp_s_bytes = rdp_bytes*n_frame;
	/// ���ɼ����껯�ĳ������֣������������꼴�ɵõ����в��������꣬ ��x_sample ��y_sample ���սǶȱ仯���У��ͺ˺���һ�£��ܹ��������ݷ��ʴ���
	// ����������r��t ������ķ�������Ϊֱ�����ɼ��������r_sample ��t_sample �������ã��������ã���ʡ�ռ䡣
	float *x_sample = (float *)malloc(rdp_bytes);
	if (*x_sample = NULL)
		printf("out of memory!");
	float *y_sample = (float *)malloc(rdp_bytes);
	if (*y_sample = NULL)
		printf("out of memory!");
	compute_x_y(x_sample, y_sample, r_step, t_size, r_N);
	/// ***************************************************************************************************************************************                              
	/// ����Ȩ�أ��������ڳ����ڴ�
	float *power = (float *)malloc(r_bytes);

	compute_power(power, r_N);



	float * radial_profile = (float *)malloc(rdp_bytes*n_frame*n_stream);  
	//float * check = (float *)malloc(rdp_bytes*n_frame*n_stream);  //�������
															// GPU allocation
	float *d_radial_profile = NULL;                                        
	//float *d_check = NULL;

	float *d_img_mat = NULL;                                                            // �豸�˴���ͼ��Ԥ��������
	float *d_xc = NULL;
	float *d_yc = NULL;
	////
	float *d_x_sample = NULL;

	float *d_y_sample = NULL;
	float *d_power = NULL;


	CHECK(hipMalloc(&d_power, r_bytes));
	CHECK(hipMalloc(&d_x_sample, rdp_bytes));
	CHECK(hipMalloc(&d_y_sample, rdp_bytes));
	CHECK(hipHostMalloc(&d_radial_profile, rdp_bytes*n_frame*n_stream));            

	CHECK(hipMalloc(&d_img_mat, img_bytes*n_frame*n_stream));                     //// �豸�˴���ͼ��Ԥ��������
	CHECK(hipMalloc(&d_xc, sizeof(float)*n_frame*n_stream));
	CHECK(hipMalloc(&d_yc, sizeof(float)*n_frame*n_stream));
	//CHECK(hipHostMalloc(&d_check, rdp_bytes*n_frame*n_stream));

	hipStream_t *streams = (hipStream_t *)malloc(n_stream * sizeof(hipStream_t));

	CHECK(hipMemcpy(d_power, power, r_bytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_x_sample, x_sample, rdp_bytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_y_sample, y_sample, rdp_bytes, hipMemcpyHostToDevice));

	/// ���������ڴ�
	// Allocate array and copy image data
	size_t pitch;


	CHECK(hipMallocPitch((void**)&d_img_mat, &pitch, arraySize * sizeof(float), arraySize*n_frame*n_stream));
	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	CHECK(hipBindTexture2D(0, &tex, d_img_mat, &desc, arraySize, arraySize*n_frame*n_stream, pitch));
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.filterMode = hipFilterModeLinear;    // --- Enable linear filtering
	tex.normalized = false;                    // --- Texture coordinates will NOT be normalized
	CHECK(hipMemcpy2D(d_img_mat, pitch, p_img_4d, sizeof(float)*arraySize, sizeof(float)*arraySize, arraySize*n_frame*n_stream, hipMemcpyHostToDevice));

	// ͼ������Ѿ��󶨵������������Ϳ����ں˺�����ʹ�������ˡ�

	for (int i = 0; i < n_stream; i++)
	{
		CHECK(hipStreamCreate(&streams[i]));
	}



	int s_rdp_LEN = r_N*t_size*n_frame;
	const dim3 block2(BLK_DIM2);
	const dim3 grid2((s_rdp_LEN + block2.x - 1) / block2.x, 1);



	// ����CPU�˵�x y ���ݣ�����Ҳ������Ҫ�����ֱ�������˵Ļ�����GPU���Ǳ������õģ�
	size_t rdpxy_byte = (r_N* 2 - 1) *n_frame*n_stream* sizeof(float);
	size_t s_rdpxy_byte = (r_N * 2 - 1) *n_frame * sizeof(float);
	float *rdp_x = (float *)malloc(rdpxy_byte);
	float *rdp_y = (float *)malloc(rdpxy_byte);

	//// ������ȡrdp_x �ķ����������η�������
	//float *rdp_x_rev = (float *)malloc(rdpxy_byte);
	//float *rdp_y_rev = (float *)malloc(rdpxy_byte);

	// ���������л���ؼ��㣬
	size_t corr_bytes = (r_N* 4 - 3) * sizeof(float);
	float *rdp_x_corr = (float*)malloc(corr_bytes);
	float *rdp_y_corr = (float*)malloc(corr_bytes);
	memset(rdp_x_corr, 0, corr_bytes);
	memset(rdp_y_corr, 0, corr_bytes);


	for (int j = 0; j < 5; j++)
	{
		//for (int i = 0; i < n_stream; i++)
		//{
		//	int s_xy_offset = i*n_frame;
		CHECK(hipMemcpy(d_xc, xc, sizeof(float)*n_frame*n_stream, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_yc, yc, sizeof(float)*n_frame*n_stream, hipMemcpyHostToDevice));
		//}
		// ���������һ�£��Ѵ���XY�ͼ�����һ��ѭ����ִ���ٶȲ��䣬������������΢�ӳ٣����ռӳ��ܵ�ִ��ʱ��

			// ����һ�·ֽ�������������GPU��CPU��ִ��ʱ��
			// ��һ������һ��GPU����
		for (int i = 0; i < n_stream; i++)
		{
			//int s_img_offset = i*n_frame*LEN;
			int s_xy_offset = i*n_frame;

			int s_rdp_offset = i*n_frame*r_N*t_size;
			compute_radial_profile_tex << <grid2, block2, 0, streams[i] >> >
				(d_xc + s_xy_offset, d_yc + s_xy_offset, d_x_sample, d_y_sample, d_power, t_size, r_N, d_radial_profile + s_rdp_offset, arraySize,s_img_offset);


		}

		for (int i = 0; i < n_stream; i++)
		{
			CHECK(hipStreamSynchronize(streams[i]));
		}

			CHECK(hipMemcpyAsync(radial_profile, d_radial_profile, rdp_bytes*n_frame*n_stream, hipMemcpyDeviceToHost, streams[0])); // ֱ��ȫ������


		//FILE *fr = fopen("r4.txt", "w");
		//if (fr == NULL)
		//{
		//	exit(1);
		//}

		//for (int t = 0; t < r_N*n_frame; t++)
		//{
		//	for (int t2 = 0; t2 < t_size; t2++)
		//	{
		//	fprintf(fr, "%f\t ",check[t2+t*t_size]);
		//	}
		//	fprintf(fr, "%\n");
		//}
		//fclose(fr);




		for (int s = 0; s < n_stream; s++)
		{

			int s_rdp_offset = s*n_frame*r_N*t_size;
			int s_xy_offset = s*n_frame;


			int s_rN_offset = s*n_frame*r_N;

			for (int i = 0; i < n_frame; i++)
			{
				int xy_offset = i;
				int rdp_offset = i*r_N*t_size;
				int rN_offset = i*r_N;
				//radial profile ��X Y�����Ϻϲ�
				getrdp_x_y(radial_profile + rdp_offset + s_rdp_offset, r_N, thetaPerQ, rdp_profile + rN_offset + s_rN_offset, rdp_x, rdp_y);


				//get_rev_rdp(rdp_x, rdp_x_rev, 2 * r_N - 1);
				//get_rev_rdp(rdp_y, rdp_y_rev, 2 * r_N - 1);

				//rdp_corr(rdp_x, rdp_x_rev, rdp_x_corr, r_N * 2 - 1);
				//rdp_corr(rdp_y, rdp_y_rev, rdp_y_corr, r_N * 2 - 1);

				self_conv(rdp_x, rdp_x_corr, 2 * r_N - 1);
				self_conv(rdp_y, rdp_y_corr, 2 * r_N - 1);


				// Ѱ�����ֵ������ֵ����������
				int pkx, pky;
				pkx = findpeak(rdp_x_corr, r_N * 2 - 1);
				pky = findpeak(rdp_y_corr, r_N * 2 - 1);


				// ��ȡ���ֵ������5���㣬���������±�
				float pkx_value[5] = { 0 };
				float pkx_index[5] = { 0 };
				float pky_value[5] = { 0 };
				float pky_index[5] = { 0 };
				for (int k = 0; k < 5; k++)
				{
					pkx_value[k] = rdp_x_corr[pkx - 2 + k];
					pky_value[k] = rdp_y_corr[pky - 2 + k];
					pkx_index[k] = (-(2 * r_N - 2) + (pkx - 2 + k))*r_step;
					pky_index[k] = (-(2 * r_N - 2) + (pky - 2 + k))*r_step;
				}

				// �������ֱ�����x y�����������߼�ֵ�㣬��������С���˷��������������

				float detx = LeastSquareGuassian(pkx_index, pkx_value, 5);
				float dety = LeastSquareGuassian(pky_index, pky_value, 5);

				//printf("the deviation of center is %f, %f\n", detx, dety);

				xc[i + s_xy_offset] = xc[i + s_xy_offset] + 2 * detx / PI;
				yc[i + s_xy_offset] = yc[i + s_xy_offset] + 2 * dety / PI;

			}
		}
	}

	// �����Ѿ������һ�ִ�ͼ��������������ȫ�����̣�ʣ�µľ��ǵ���QI��
	//printf("good until now!\n");

	for (int i = 0; i < n_stream; i++)
	{
		CHECK(hipStreamDestroy(streams[i]));
	}

	//for (int i = 0; i < n_stream; i++)
	//{
	//	int xy_offset = i*n_frame;
	//	for (int j = 0; j < n_frame; j++)
	//	{
	//		printf("the real center of image is %f, %f\n", xc[j+xy_offset], yc[j+xy_offset]);
	//	}

	//}


	/////// �ͷ��ڴ�ռ�


	//free(check);
	//check = NULL;

	// �ͷ�������ڴ�

	free(x_sample);
	x_sample = NULL;
	free(y_sample);
	y_sample = NULL;


	free(radial_profile);
	radial_profile = NULL;
	free(rdp_x);
	rdp_x = NULL;
	free(rdp_y);
	rdp_y = NULL;
	free(power);
	power = NULL;
	//free(rdp_x_rev);
	//rdp_x_rev = NULL;
	//free(rdp_y_rev);
	//rdp_y_rev = NULL;
	free(rdp_x_corr);
	rdp_x_corr = NULL;
	free(rdp_y_corr);
	rdp_y_corr = NULL;



	////
	////
	//CHECK(hipHostFree(d_check));
	CHECK(hipHostFree(d_radial_profile));


	CHECK(hipFree(d_img_mat));
	CHECK(hipFree(d_xc));
	CHECK(hipFree(d_yc));
	CHECK(hipFree(d_x_sample));
	CHECK(hipFree(d_y_sample));
	CHECK(hipFree(d_power));

}


