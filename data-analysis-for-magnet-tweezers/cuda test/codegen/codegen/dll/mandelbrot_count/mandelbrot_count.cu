#include "hip/hip_runtime.h"
/*
 * File: mandelbrot_count.cu
 *
 * GPU Coder version                    : 1.0
 * CUDA/C/C++ source code generated on  : 26-Dec-2017 22:57:54
 */

/* Include Files */
#include "rt_nonfinite.h"
#include "mandelbrot_count.h"

/* Function Declarations */
static __global__ void mandelbrot_count_kernel1(const real_T *yGrid, const
  real_T *xGrid, creal_T *z, real_T *count, creal_T *z0);
static __global__ void mandelbrot_count_kernel2(real_T *count);
static __global__ void mandelbrot_count_kernel3(creal_T *z0, real_T *count,
  real_T *y, creal_T *z);
static __device__ real_T rt_hypotd_snf(real_T u0, real_T u1);

/* Function Definitions */

/*
 * Arguments    : uint3 blockArg
 *                uint3 gridArg
 *                const real_T *yGrid
 *                const real_T *xGrid
 *                creal_T *z
 *                real_T *count
 *                creal_T *z0
 * Return Type  : void
 */
static __global__ __launch_bounds__(512, 1) void mandelbrot_count_kernel1(const
  real_T *yGrid, const real_T *xGrid, creal_T *z, real_T *count, creal_T *z0)
{
  int32_T k;
  ;
  ;
  k = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if (!(k >= 1000000)) {
    /*  Add Kernelfun pragma to trigger kernel creation */
    z0[k].re = xGrid[k] + 0.0 * yGrid[k];
    z0[k].im = yGrid[k];
    count[k] = 1.0;
    z[k] = z0[k];
  }
}

/*
 * Arguments    : uint3 blockArg
 *                uint3 gridArg
 *                real_T *count
 * Return Type  : void
 */
static __global__ __launch_bounds__(512, 1) void mandelbrot_count_kernel2(real_T
  *count)
{
  int32_T k;
  ;
  ;
  k = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if (!(k >= 1000000)) {
    count[k] = log(count[k]);
  }
}

/*
 * Arguments    : uint3 blockArg
 *                uint3 gridArg
 *                creal_T *z0
 *                real_T *count
 *                real_T *y
 *                creal_T *z
 * Return Type  : void
 */
static __global__ __launch_bounds__(512, 1) void mandelbrot_count_kernel3
  (creal_T *z0, real_T *count, real_T *y, creal_T *z)
{
  real_T z_im;
  int32_T k;
  ;
  ;
  k = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if (!(k >= 1000000)) {
    z_im = z[k].re * z[k].im + z[k].im * z[k].re;
    z[k].re = (z[k].re * z[k].re - z[k].im * z[k].im) + z0[k].re;
    z[k].im = z_im + z0[k].im;
    y[k] = rt_hypotd_snf(z[k].re, z[k].im);
    count[k] += (real_T)(y[k] <= 2.0);
  }
}

/*
 * Arguments    : real_T u0
 *                real_T u1
 * Return Type  : real_T
 */
static __device__ real_T rt_hypotd_snf(real_T u0, real_T u1)
{
  real_T y;
  real_T a;
  real_T b;
  a = fabs(u0);
  b = fabs(u1);
  if (a < b) {
    a /= b;
    y = b * sqrt(a * a + 1.0);
  } else if (a > b) {
    b /= a;
    y = a * sqrt(b * b + 1.0);
  } else if (isnan(b)) {
    y = b;
  } else {
    y = a * 1.4142135623730951;
  }

  return y;
}

/*
 * mandelbrot computation
 * Arguments    : real_T maxIterations
 *                const real_T xGrid[1000000]
 *                const real_T yGrid[1000000]
 *                real_T count[1000000]
 * Return Type  : void
 */
void mandelbrot_count(real_T maxIterations, const real_T xGrid[1000000], const
                      real_T yGrid[1000000], real_T count[1000000])
{
  int32_T n;
  real_T *gpu_yGrid;
  real_T *gpu_xGrid;
  creal_T *gpu_z;
  real_T *gpu_count;
  creal_T *gpu_z0;
  real_T *gpu_y;
  hipMalloc(&gpu_y, 8000000ULL);
  hipMalloc(&gpu_z0, 16000000ULL);
  hipMalloc(&gpu_z, 16000000ULL);
  hipMalloc(&gpu_count, 8000000ULL);
  hipMalloc(&gpu_xGrid, 8000000ULL);
  hipMalloc(&gpu_yGrid, 8000000ULL);

  /*  Add Kernelfun pragma to trigger kernel creation */
  hipMemcpy((void *)gpu_yGrid, (void *)&yGrid[0], 8000000ULL,
             hipMemcpyHostToDevice);
  hipMemcpy((void *)gpu_xGrid, (void *)&xGrid[0], 8000000ULL,
             hipMemcpyHostToDevice);
  mandelbrot_count_kernel1<<<dim3(1954U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (gpu_yGrid, gpu_xGrid, gpu_z, gpu_count, gpu_z0);
  for (n = 0; n < (int32_T)(maxIterations + 1.0); n++) {
    mandelbrot_count_kernel3<<<dim3(1954U, 1U, 1U), dim3(512U, 1U, 1U)>>>(gpu_z0,
      gpu_count, gpu_y, gpu_z);
  }

  mandelbrot_count_kernel2<<<dim3(1954U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (gpu_count);
  hipMemcpy((void *)&count[0], (void *)gpu_count, 8000000ULL,
             hipMemcpyDeviceToHost);
  hipFree(gpu_yGrid);
  hipFree(gpu_xGrid);
  hipFree(gpu_count);
  hipFree(gpu_z);
  hipFree(gpu_z0);
  hipFree(gpu_y);
}

/*
 * File trailer for mandelbrot_count.cu
 *
 * [EOF]
 */
