#include "hip/hip_runtime.h"
/*
 * File: cuda_1.cu
 *
 * GPU Coder version                    : 1.0
 * CUDA/C/C++ source code generated on  : 23-Jul-2018 11:04:01
 */

/* Include Files */
#include "rt_nonfinite.h"
#include "cuda_1.h"

/* Function Declarations */
static __global__ void cuda_1_kernel1(const real_T *x, real_T *y_data);

/* Function Definitions */

/*
 * Arguments    : uint3 blockArg
 *                uint3 gridArg
 *                const real_T *x
 *                real_T *y_data
 * Return Type  : void
 */
static __global__ __launch_bounds__(512, 1) void cuda_1_kernel1(const real_T *x,
  real_T *y_data)
{
  uint32_T threadId;
  int32_T i;
  int32_T j;
  ;
  ;
  threadId = ((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y) +
                blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
               threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y * blockDim.x)
    + threadIdx.x;
  i = (int32_T)(threadId / 1000U);
  j = (int32_T)(threadId - (uint32_T)i * 1000U);
  if ((!(j >= 1000)) && (!(i >= 1000))) {
    y_data[i + 1000 * j] = x[i + 1000 * j] * x[i + 1000 * j] / ((real_T)(i + j)
      + 2.0);
  }
}

/*
 * Arguments    : const real_T x[1000000]
 *                real_T n
 *                real_T y_data[]
 *                int32_T y_size[2]
 * Return Type  : void
 */
void cuda_1(const real_T x[1000000], real_T n, real_T y_data[], int32_T y_size[2])
{
  int32_T j;
  int32_T i;
  real_T *gpu_x;
  real_T *gpu_y_data;
  boolean_T y_data_dirtyOnGpu;
  hipMalloc(&gpu_y_data, 1000000U * sizeof(real_T));
  hipMalloc(&gpu_x, 8000000ULL);
  y_data_dirtyOnGpu = false;

  /*  ���в��Ե�һ�֣����÷�������GPU�Ƿ��������� */
  y_size[0] = 1;
  y_size[1] = 1;
  y_data[0] = 0.0;
  if (n == 1.0) {
    y_size[0] = 1000;
    y_size[1] = 1000;
    for (j = 0; j < 1000000; j++) {
      y_data[j] = 0.0;
    }

    for (i = 0; i < 1000; i++) {
      for (j = 0; j < 1000; j++) {
        y_data[i + 1000 * j] = x[i + 1000 * j] * x[i + 1000 * j] / ((real_T)(i +
          j) + 2.0);
      }
    }
  } else {
    if (n == 2.0) {
      y_size[0] = 1000;
      y_size[1] = 1000;
      hipMemcpy((void *)gpu_x, (void *)&x[0], 8000000ULL,
                 hipMemcpyHostToDevice);
      hipMemcpy((void *)gpu_y_data, (void *)&y_data[0], 1000000U * sizeof
                 (real_T), hipMemcpyHostToDevice);
      cuda_1_kernel1<<<dim3(1954U, 1U, 1U), dim3(512U, 1U, 1U)>>>(gpu_x,
        gpu_y_data);
      y_data_dirtyOnGpu = true;
    }
  }

  if (y_data_dirtyOnGpu) {
    hipMemcpy((void *)&y_data[0], (void *)gpu_y_data, y_size[0] * y_size[1] *
               sizeof(real_T), hipMemcpyDeviceToHost);
  }

  hipFree(gpu_x);
  hipFree(gpu_y_data);
}

/*
 * File trailer for cuda_1.cu
 *
 * [EOF]
 */
