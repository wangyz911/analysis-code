#include "hip/hip_runtime.h"
/*
 * cuda_1.cu
 *
 * Code generation for function 'cuda_1'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "cuda_1.h"
#include "cuda_1_data.h"

/* Function Declarations */
static void checkCudaError(hipError_t errCode, const char * file, uint32_T line);
static __global__ void cuda_1_kernel1(const real_T *x, real_T *y_data);

/* Function Definitions */
static void checkCudaError(hipError_t errCode, const char * file, uint32_T line)
{
  emlrtRTEInfo rtInfo;
  char_T *pn;
  char_T *fn;
  char_T *brk;
  uint32_T len;
  if (errCode != hipSuccess) {
    len = strlen(file);
    pn = (char_T *)calloc(len + 1U, 1U);
    fn = (char_T *)calloc(len + 1U, 1U);
    memcpy(pn, file, len);
    memcpy(fn, file, len);
    brk = strrchr(fn, '.');
    *brk = '\x00';
    brk = strrchr(fn, '/');
    if (brk == NULL) {
      brk = strrchr(fn, '\\');
    }

    brk++;
    rtInfo.lineNo = (int32_T)line;
    rtInfo.colNo = (int32_T)0U;
    rtInfo.fName = brk;
    rtInfo.pName = pn;
    emlrtCUDAError(errCode, hipGetErrorName(errCode), hipGetErrorString
                   (errCode), &rtInfo, emlrtRootTLSGlobal);
  }
}

static __global__ __launch_bounds__(512, 1) void cuda_1_kernel1(const real_T *x,
  real_T *y_data)
{
  uint32_T threadId;
  int32_T i;
  int32_T j;
  ;
  ;
  threadId = ((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y) +
                blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
               threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y * blockDim.x)
    + threadIdx.x;
  i = (int32_T)(threadId / 1000U);
  j = (int32_T)(threadId - (uint32_T)i * 1000U);
  if ((!(j >= 1000)) && (!(i >= 1000))) {
    y_data[i + 1000 * j] = x[i + 1000 * j] * x[i + 1000 * j] / ((real_T)(i + j)
      + 2.0);
  }
}

void cuda_1(const real_T x[1000000], real_T n, real_T y_data[], int32_T y_size[2])
{
  int32_T j;
  int32_T i;
  real_T *gpu_x;
  real_T *gpu_y_data;
  boolean_T y_data_dirtyOnGpu;

#define CUDACHECK(errCall)             checkCudaError(errCall, __FILE__, __LINE__)

  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  CUDACHECK(hipMalloc(&gpu_y_data, 1000000U * sizeof(real_T)));
  CUDACHECK(hipMalloc(&gpu_x, 8000000ULL));
  y_data_dirtyOnGpu = false;

  /*  ���в��Ե�һ�֣����÷�������GPU�Ƿ��������� */
  y_size[0] = 1;
  y_size[1] = 1;
  y_data[0] = 0.0;
  if (n == 1.0) {
    y_size[0] = 1000;
    y_size[1] = 1000;
    for (j = 0; j < 1000000; j++) {
      y_data[j] = 0.0;
    }

    for (i = 0; i < 1000; i++) {
      for (j = 0; j < 1000; j++) {
        y_data[i + 1000 * j] = x[i + 1000 * j] * x[i + 1000 * j] / ((real_T)(i +
          j) + 2.0);
      }
    }
  } else {
    if (n == 2.0) {
      y_size[0] = 1000;
      y_size[1] = 1000;
      CUDACHECK(hipMemcpy((void *)gpu_x, (void *)&x[0], 8000000ULL,
                           hipMemcpyHostToDevice));
      CUDACHECK(hipMemcpy((void *)gpu_y_data, (void *)&y_data[0], 1000000U *
                           sizeof(real_T), hipMemcpyHostToDevice));
      cuda_1_kernel1<<<dim3(1954U, 1U, 1U), dim3(512U, 1U, 1U)>>>(gpu_x,
        gpu_y_data);
      CUDACHECK(hipGetLastError());
      y_data_dirtyOnGpu = true;
    }
  }

  if (y_data_dirtyOnGpu) {
    CUDACHECK(hipMemcpy((void *)&y_data[0], (void *)gpu_y_data, y_size[0] *
                         y_size[1] * sizeof(real_T), hipMemcpyDeviceToHost));
  }

  CUDACHECK(hipFree(gpu_x));
  CUDACHECK(hipFree(gpu_y_data));

#undef CUDACHECK

}

/* End of code generation (cuda_1.cu) */
